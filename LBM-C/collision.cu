#include "hip/hip_runtime.h"
#ifndef COLLISION
#define COLLISION

// Necessary includes
#include "macros.cu"
#include "collision.cuh"
#include "solver.cu"

// These files are only included to remove squiggly red lines in VS2010
#include "data_types.cuh"
#include "hip/hip_runtime.h"

// THIS DECLARATION *SHOULD* BE IN solver.cu FOR CLARITY, HOWEVER COMPILER COMPLAINS IF THIS IS THE CASE
__device__ __constant__ DomainConstant domain_constants;

#define POW4(x) x*x*x*x
#define INVERSEPOW(x) {1./x}

__device__ collision collision_functions[9] = { bgk_collision, bgk_guo_collision, bgk_ntpor_collision, bgk_ntpor_guo_collision,
												mrt_collision, mrt_guo_collision, mrt_ntpor_collision, mrt_ntpor_guo_collision,
												bounceback};

__device__ inline double u_square(Node *current_node)
{
	double value = 0;

	#pragma unroll
	for(int d = 0; d<DIM; d++)
	{
		value += (current_node->u[d]*current_node->u[d]);
	}

	return value*1.5;
}

__device__ inline double e_mul_u(Node *current_node, int *i)
{
	double value = 0;

	#pragma unroll
	for(int d = 0; d<DIM; d++)
	{
		value += (domain_constants.e[d][*i]*current_node->u[d]);
	}

	return value*3.;
}

__device__ __noinline__ void bgk_collision(Node *current_node, double *tau)
{
	double f_eq[Q], u_sq, eu;

	u_sq = u_square(current_node);

	for(int i=0;i<Q;i++)
	{
		eu = e_mul_u(current_node, &i);
		f_eq[i] = current_node->rho*domain_constants.omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	}

	if (current_node->c_smag>0) turbulent_viscosity(current_node, f_eq, tau);

	for(int i = 0; i<Q; i++)
	{
		current_node->f[i] = current_node->f[i] - (1.0/(*tau)) * (current_node->f[i]-f_eq[i]);
	}
}

__device__ __noinline__ void bgk_guo_collision(Node *current_node, double *tau)
{
	double f_eq[Q], u_sq, eu, F_coeff[DIM], force_term[Q];
	int d;
	
	#pragma unroll
	for(d = 0; d<DIM; d++)
	{
		current_node->u[d] = current_node->u[d] + (1/(2*current_node->rho))*current_node->F[d];
	}

	u_sq = u_square(current_node);

	for(int i=0;i<Q;i++)
	{
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
		#if DIM > 2
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1])+(domain_constants.e[2][i]*current_node->u[2]))));
		#else
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1]))));
		#endif
		}

		force_term[i] = 0;
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
			force_term[i] += F_coeff[d]*current_node->F[d];
		}

		eu = e_mul_u(current_node, &i);
		f_eq[i] = current_node->rho*domain_constants.omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	}

	if (current_node->c_smag>0) turbulent_viscosity(current_node, f_eq, tau);

	for(int i=0;i<Q;i++)
	{
		current_node->f[i] = current_node->f[i] - (1.0/(*tau)) * (current_node->f[i]-f_eq[i]) + force_term[i];
	}
}

__device__ __noinline__ void bgk_ntpor_collision(Node *current_node, double *tau)
{
	double f_eq[Q], u_sq, eu, collision_bgk, collision_s, tmp[Q];

	u_sq = u_square(current_node);
	for(int i=0;i<Q;i++)
	{
		eu = e_mul_u(current_node, &i);
		f_eq[i] = current_node->rho*domain_constants.omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	}

	if (current_node->c_smag>0) turbulent_viscosity(current_node, f_eq, tau);
	
	for(int i =0;i<Q;i++)
	{
		collision_bgk = (1.0/(*tau)) * (current_node->f[i]-f_eq[i]);
		collision_s = current_node->f[domain_constants.opp[i]]-current_node->f[i];
		tmp[i] = current_node->f[i] - (1-(current_node->B))*collision_bgk + (current_node->B)*collision_s;
	}

	for(int i =0;i<Q;i++)
	{
		current_node->f[i] = tmp[i];
	}

}

__device__ void bgk_ntpor_guo_collision(Node *current_node, double *tau)
{
	double f_eq[Q], u_sq, eu, collision_bgk, collision_s, F_coeff[DIM], force_term[Q], tmp[Q];
	int d;

	#pragma unroll
	for(d = 0; d<DIM; d++)
	{
		current_node->u[d] = current_node->u[d] + (1/(2*current_node->rho))*current_node->F[d];
	}

	u_sq = u_square(current_node);

	for(int i=0;i<Q;i++)
	{
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
		#if DIM > 2
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1])+(domain_constants.e[2][i]*current_node->u[2]))));
		#else
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1]))));
		#endif
		}

		force_term[i] = 0;
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
			force_term[i] += F_coeff[d]*current_node->F[d];
		}

		eu = e_mul_u(current_node, &i);
		f_eq[i] = current_node->rho*domain_constants.omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	}

	if (current_node->c_smag>0) turbulent_viscosity(current_node, f_eq, tau);

	for(int i =0;i<Q;i++)
	{
		collision_bgk = (1.0/(*tau)) * (current_node->f[i]-f_eq[i]);
		collision_s = current_node->f[domain_constants.opp[i]]-current_node->f[i];

		tmp[i] = current_node->f[i] - (1-(current_node->B))*(collision_bgk) + (current_node->B)*collision_s + (1-(current_node->B))*force_term[i];
	}

	for(int i =0;i<Q;i++)
	{
		current_node->f[i] = tmp[i];
	}
}

__device__ void mrt_collision(Node *current_node, double *tau)
{
	double m_eq[Q],m[Q];
	
	#ifdef D2Q9
		meq_d2q9(current_node,m_eq);
	#endif

	#ifdef D3Q15
		meq_d3q15(current_node, m_eq);
	#endif

	for(int i = 0; i<Q; i++)
	{
		m[i] = 0;
		for(int j=0; j<Q; j++)
		{
			m[i] = m[i] + domain_constants.M[i][j]*current_node->f[j];
		}
	}

	for(int i = 0; i<Q;i++)
	{
		m[i] = domain_constants.tau_mrt[i]*(m[i]-m_eq[i]);
	}

	for(int i = 0; i<Q; i++)
	{
		//reuse m_eq to save on memory...
		m_eq[i] = 0;
		for(int j=0; j<Q; j++)
		{
			
			m_eq[i] = m_eq[i] + domain_constants.M_inv[i][j]*m[j];
		}
		current_node->f[i] = current_node->f[i] - m_eq[i]; // m_eq here is not equilibrium distribution, 
	}													   // it is the result of previous computation!!!!
	
}

__device__ void mrt_guo_collision(Node *current_node, double *tau)
{
	double m_eq[Q],m[Q], F_coeff[DIM], force_term[Q];
	int d;
	
	// Add force contribution to velocity
	#pragma unroll
	for(d = 0; d<DIM; d++)
	{
		current_node->u[d] = current_node->u[d] + (1/(2*current_node->rho))*current_node->F[d];
	}

	// Calculate forcing term
	for(int i=0;i<Q;i++)
	{
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
		#if DIM > 2
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1])+(domain_constants.e[2][i]*current_node->u[2]))));
		#else
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1]))));
		#endif
		}

		force_term[i] = 0;
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
			force_term[i] += F_coeff[d]*current_node->F[d];
		}
	}

	// Calculate equilibrium distribution
	#ifdef D2Q9
		meq_d2q9(current_node,m_eq);
	#endif

	#ifdef D3Q15
		meq_d3q15(current_node, m_eq);
	#endif

	// Convert PDF's to MDF's (Momentum Distribution Function)
	for(int i = 0; i<Q; i++)
	{
		m[i] = 0;
		for(int j=0; j<Q; j++)
		{
			m[i] = m[i] + domain_constants.M[i][j]*current_node->f[j];
		}
	}

	// Execute MRT collision
	for(int i = 0; i<Q;i++)
	{
		m[i] = domain_constants.tau_mrt[i]*(m[i]-m_eq[i]);
	}

	// convert MDF's back to PDF's adding the result of collision and forcing
	for(int i = 0; i<Q; i++)
	{
		//reuse m_eq to save on memory...
		m_eq[i] = 0;
		for(int j=0; j<Q; j++)
		{
			
			m_eq[i] = m_eq[i] + domain_constants.M_inv[i][j]*m[j];
		}
		current_node->f[i] = current_node->f[i] - m_eq[i] + force_term[i]; // m_eq here is not equilibrium distribution, 
	}																		// it is the result of previous computation!!!!
	
}

__device__ void mrt_ntpor_collision(Node *current_node, double *tau)
{
	double m_eq[Q],m[Q], collision_s;
	
	#ifdef D2Q9
		meq_d2q9(current_node,m_eq);
	#endif

	#ifdef D3Q15
		meq_d3q15(current_node, m_eq);
	#endif

	for(int i = 0; i<Q; i++)
	{
		m[i] = 0;
		for(int j=0; j<Q; j++)
		{
			m[i] = m[i] + domain_constants.M[i][j]*current_node->f[j];
		}
	}

	for(int i = 0; i<Q;i++)
	{
		m[i] = domain_constants.tau_mrt[i]*(m[i]-m_eq[i]);
	}

	for(int i = 0; i<Q; i++)
	{
		//reuse m_eq to save on memory...
		m_eq[i] = 0;
		for(int j=0; j<Q; j++)
		{
			m_eq[i] = m_eq[i] + domain_constants.M_inv[i][j]*m[j];
		}

		collision_s = current_node->f[domain_constants.opp[i]]-current_node->f[i];

		current_node->f[i] = current_node->f[i] - (1-(current_node->B))*m_eq[i] + (current_node->B)*collision_s; // m_eq here is not equilibrium distribution, 
	}													   // it is the result of previous computation!!!!
	
}

__device__ void mrt_ntpor_guo_collision(Node *current_node, double *tau)
{
	double m_eq[Q],m[Q], F_coeff[DIM], force_term[Q], collision_s;
	int d;
	
	// Add force contribution to velocity
	#pragma unroll
	for(d = 0; d<DIM; d++)
	{
		current_node->u[d] = current_node->u[d] + (1/(2*current_node->rho))*current_node->F[d];
	}

	// Calculate forcing term
	for(int i=0;i<Q;i++)
	{
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
		#if DIM > 2
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1])+(domain_constants.e[2][i]*current_node->u[2]))));
		#else
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1]))));
		#endif
		}

		force_term[i] = 0;
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
			force_term[i] += F_coeff[d]*current_node->F[d];
		}
	}

	// Calculate equilibrium distribution
	#ifdef D2Q9
		meq_d2q9(current_node,m_eq);
	#endif

	#ifdef D3Q15
		meq_d3q15(current_node, m_eq);
	#endif

	// Convert PDF's to MDF's (Momentum Distribution Function)
	for(int i = 0; i<Q; i++)
	{
		m[i] = 0;
		for(int j=0; j<Q; j++)
		{
			m[i] = m[i] + domain_constants.M[i][j]*current_node->f[j];
		}
	}

	// Execute MRT collision
	for(int i = 0; i<Q;i++)
	{
		m[i] = domain_constants.tau_mrt[i]*(m[i]-m_eq[i]);
	}

	// convert MDF's back to PDF's adding the result of collision and forcing
	for(int i = 0; i<Q; i++)
	{
		//reuse m_eq to save on memory...
		m_eq[i] = 0;
		for(int j=0; j<Q; j++)
		{
			
			m_eq[i] = m_eq[i] + domain_constants.M_inv[i][j]*m[j];
		}

		collision_s = current_node->f[domain_constants.opp[i]]-current_node->f[i];

		current_node->f[i] = current_node->f[i] - (1-(current_node->B))*m_eq[i] + (1-(current_node->B))*force_term[i] + (current_node->B)*collision_s; // m_eq here is not equilibrium distribution, 
	}																		// it is the result of previous computation!!!!
	
}

__device__ void bounceback(Node *current_node, double *tau)
{
	double tmp[Q];
	for(int i=0;i<Q;i++)
	{
		tmp[i] = current_node->f[i];
	}

	for(int i=0;i<Q;i++)
	{
		current_node->f[i] = tmp[domain_constants.opp[i]];
	}

	current_node->u[0] = 0;
	current_node->u[1] = 0;
	#if DIM > 2
		current_node->u[2] = 0;
	#endif

	current_node->rho = 0;
}

__device__ void turbulent_viscosity(Node *current_node, double *f_eq, double *tau)
{
	double q_bar[DIM][DIM];
	double q_hat = 0.;

	for(int i = 0; i<DIM; i++)
	{
		for(int j = 0; j<DIM; j++)
		{
			for(int q = 0; q<Q; q++)
			{
				q_bar[i][j] = q_bar[i][j]+((double)domain_constants.e[i][q]*(double)domain_constants.e[j][q]*(current_node->f[q]-f_eq[q]));
			}
			q_hat = q_hat + sqrt((double)2*q_bar[i][j]*q_bar[i][j]);
		}
	}
	
	*tau = *tau+0.5*(sqrt(((*tau)*(*tau))+(2*sqrt((double)2)*(current_node->c_smag*current_node->c_smag)*(1/(current_node->rho*POW4(1/sqrt((double)3))))*q_hat))-*tau);
}

__device__ void meq_d2q9(Node *current_node, double *meq)
{
	double jx = current_node->rho*current_node->u[0];
	double jy = current_node->rho*current_node->u[1];

	meq[0] = current_node->rho;
	meq[1] = (-2*current_node->rho)+3*(jx*jx+jy*jy);
	meq[2] = current_node->rho-3*(jx*jx+jy*jy);
	meq[3] = jx;
	meq[4] = -jx;
	meq[5] = jy;
	meq[6] = -jy;
	meq[7] = jx*jx-jy*jy;
	meq[8] = jx*jy;
}

__device__ void meq_d3q15(Node *current_node, double *meq)
{
	double jx = current_node->rho*current_node->u[0];
	double jy = current_node->rho*current_node->u[1];
	double jz = current_node->rho*current_node->u[2];

	meq[0] = current_node->rho;
	meq[1] = (-1*current_node->rho)+(jx*jx+jy*jy+jz*jz);
	//meq[2] = -current_node->rho;
	meq[2] = current_node->rho-5*(jx*jx+jy*jy+jz*jz);
	meq[3] = jx;
	meq[4] = (-7/3)*jx;
	meq[5] = jy;
	meq[6] = (-7/3)*jy;
	meq[7] = jz;
	meq[8] = (-7/3)*jz;
	meq[9] = (2*jx*jx-(jy*jy+jz*jz));
	meq[10] = jy*jy-jz*jz;
	meq[11] = jx*jy;
	meq[12] = jy*jz;
	meq[13] = jx*jz;
	meq[14] = 0;
}
#endif
