#include "hip/hip_runtime.h"
#ifndef COLLISION
#define COLLISION

// Necessary includes
#include "macros.cu"
#include "collision.cuh"
#include "solver.cu"

// These files are only included to remove squiggly red lines in VS2010
#include "data_types.cuh"
#include "hip/hip_runtime.h"

// THIS DECLARATION *SHOULD* BE IN solver.cu FOR CLARITY, HOWEVER COMPILER COMPLAINS IF THIS IS THE CASE
__device__ __constant__ DomainConstant domain_constants;

#define POW4(x) x*x*x*x
#define INVERSEPOW(x) {1./x}

__device__ collision collision_functions[5] = { bgk_collision, guo_bgk_collision, ntpor_collision, guo_ntpor_collision, bounceback};

__device__ inline double u_square(Node *current_node)
{
	double value = 0;

	#pragma unroll
	for(int d = 0; d<DIM; d++)
	{
		value += (current_node->u[d]*current_node->u[d]);
	}

	return value*1.5;
}

__device__ inline double e_mul_u(Node *current_node, int *i)
{
	double value = 0;

	#pragma unroll
	for(int d = 0; d<DIM; d++)
	{
		value += (domain_constants.e[d][*i]*current_node->u[d]);
	}

	return value*3.;
}

__device__ __noinline__ void bgk_collision(Node *current_node, double *tau)
{
	double f_eq[Q], u_sq, eu;

	u_sq = u_square(current_node);
	for(int i=0;i<Q;i++)
	{
		eu = e_mul_u(current_node, &i);
		f_eq[i] = current_node->rho*domain_constants.omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	}

	if (current_node->c_smag>0) turbulent_viscosity(current_node, f_eq, tau);

	for(int i = 0; i<Q; i++)
	{
		current_node->f[i] = current_node->f[i] - (1.0/(*tau)) * (current_node->f[i]-f_eq[i]);
	}
}

__device__ __noinline__ void guo_bgk_collision(Node *current_node, double *tau)
{
	double f_eq[Q], u_sq, eu, F_coeff[DIM], force_term[Q];
	int d;
	
	#pragma unroll
	for(d = 0; d<DIM; d++)
	{
		current_node->u[d] = current_node->u[d] + (1/2)*current_node->rho*current_node->F[d];
	}

	u_sq = u_square(current_node);

	for(int i=0;i<Q;i++)
	{
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
		#if DIM > 2
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1])+(domain_constants.e[2][i]*current_node->u[2]))));
		#else
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1]))));
		#endif
		}

		force_term[i] = 0;
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
			force_term[i] += F_coeff[d]*current_node->F[d];
		}

		eu = e_mul_u(current_node, &i);
		f_eq[i] = current_node->rho*domain_constants.omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	}

	if (current_node->c_smag>0) turbulent_viscosity(current_node, f_eq, tau);

	for(int i=0;i<Q;i++)
	{
		current_node->f[i] = current_node->f[i] - (1.0/(*tau)) * (current_node->f[i]-f_eq[i]) + force_term[i];
	}
}

__device__ __noinline__ void ntpor_collision(Node *current_node, double *tau)
{
	double f_eq[Q], u_sq, eu, collision_bgk, collision_s, tmp[Q];

	u_sq = u_square(current_node);
	for(int i=0;i<Q;i++)
	{
		eu = e_mul_u(current_node, &i);
		f_eq[i] = current_node->rho*domain_constants.omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	}

	if (current_node->c_smag>0) turbulent_viscosity(current_node, f_eq, tau);
	
	for(int i =0;i<Q;i++)
	{
		collision_bgk = (1.0/(*tau)) * (current_node->f[i]-f_eq[i]);
		collision_s = current_node->f[domain_constants.opp[i]]-current_node->f[i];
		tmp[i] = current_node->f[i] - (1-(current_node->B))*collision_bgk + (current_node->B)*collision_s;
	}

	for(int i =0;i<Q;i++)
	{
		current_node->f[i] = tmp[i];
	}

}

__device__ void guo_ntpor_collision(Node *current_node, double *tau)
{
	double f_eq[Q], u_sq, eu, collision_bgk, collision_s, F_coeff[DIM], force_term[Q], tmp[Q];
	int d;

	#pragma unroll
	for(d = 0; d<DIM; d++)
	{
		current_node->u[d] = current_node->u[d] + (1/2)*current_node->rho*current_node->F[d];
	}

	u_sq = u_square(current_node);

	for(int i=0;i<Q;i++)
	{
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
		#if DIM > 2
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1])+(domain_constants.e[2][i]*current_node->u[2]))));
		#else
			F_coeff[d] = domain_constants.omega[i]*(1-(1/(2*(*tau))))*(((domain_constants.e[d][i]-current_node->u[d])*3)+(domain_constants.e[d][i]*9*((domain_constants.e[0][i]*current_node->u[0])+(domain_constants.e[1][i]*current_node->u[1]))));
		#endif
		}

		force_term[i] = 0;
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
			force_term[i] += F_coeff[d]*current_node->F[d];
		}

		eu = e_mul_u(current_node, &i);
		f_eq[i] = current_node->rho*domain_constants.omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	}

	if (current_node->c_smag>0) turbulent_viscosity(current_node, f_eq, tau);

	for(int i =0;i<Q;i++)
	{
		collision_bgk = (1.0/(*tau)) * (current_node->f[i]-f_eq[i]);
		collision_s = current_node->f[domain_constants.opp[i]]-current_node->f[i];

		tmp[i] = current_node->f[i] - (1-(current_node->B))*(collision_bgk) + (current_node->B)*collision_s + (1-(current_node->B))*force_term[i];
	}

	for(int i =0;i<Q;i++)
	{
		current_node->f[i] = tmp[i];
	}
}

__device__ void bounceback(Node *current_node, double *tau)
{
	double tmp[Q];
	for(int i=0;i<Q;i++)
	{
		tmp[i] = current_node->f[i];
	}

	for(int i=0;i<Q;i++)
	{
		current_node->f[i] = tmp[domain_constants.opp[i]];
	}

	current_node->u[0] = 0;
	current_node->u[1] = 0;
	current_node->rho = 0;
}

__device__ void turbulent_viscosity(Node *current_node, double *f_eq, double *tau)
{
	double q_bar[DIM][DIM];
	double q_hat = 0.;

	for(int i = 0; i<DIM; i++)
	{
		for(int j = 0; j<DIM; j++)
		{
			for(int q = 0; q<Q; q++)
			{
				q_bar[i][j] = q_bar[i][j]+((double)domain_constants.e[i][q]*(double)domain_constants.e[j][q]*(current_node->f[q]-f_eq[q]));
			}
			q_hat = q_hat + sqrt((double)2*q_bar[i][j]*q_bar[i][j]);
		}
	}
	
	*tau = *tau+0.5*(sqrt(((*tau)*(*tau))+(2*sqrt((double)2)*(current_node->c_smag*current_node->c_smag)*(1/(current_node->rho*POW4(1/sqrt((double)3))))*q_hat))-*tau);
}

#endif
