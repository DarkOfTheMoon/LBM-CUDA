#ifndef COLLISION
#define COLLISION

// Necessary includes
#include "macros.cu"
#include "collision.cuh"

// These files are only included to remove squiggly red lines in VS2010
#include "data_types.cuh"
#include "hip/hip_runtime.h"

__device__ collision collision_functions[5] = { bgk_collision, guo_bgk_collision, ntpor_collision, guo_ntpor_collision, bounceback};

__device__ inline double u_square(Node *current_node)
{
	double value = 0;

	#pragma unroll
	for(int d = 0; d<DIM; d++)
	{
		value += (current_node->u[d]*current_node->u[d]);
	}

	return value*1.5;
}

__device__ inline double e_mul_u(Node *current_node, int e[DIM][Q], int *i)
{
	double value = 0;

	#pragma unroll
	for(int d = 0; d<DIM; d++)
	{
		value += (e[d][*i]*current_node->u[d]);
	}

	return value*3.;
}

__device__ __noinline__ void bgk_collision(Node *current_node, int *opp, int e[DIM][Q], double *omega, double *tau, double *B)
{
	double f_eq, u_sq, eu;

	u_sq = u_square(current_node);
	for(int i=0;i<Q;i++)
	{
		eu = e_mul_u(current_node, e, &i);
		f_eq = current_node->rho*omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	
		current_node->f[i] = current_node->f[i] - (1.0/(*tau)) * (current_node->f[i]-f_eq);
	}
}

__device__ __noinline__ void guo_bgk_collision(Node *current_node, int *opp, int e[DIM][Q], double *omega, double *tau, double *B)
{
	double f_eq, u_sq, eu, F_coeff[DIM], force_term;
	int d;
	
	#pragma unroll
	for(d = 0; d<DIM; d++)
	{
		current_node->u[d] = current_node->u[d] + (1/2)*current_node->rho*current_node->F[d];
	}

	u_sq = u_square(current_node);

	for(int i=0;i<Q;i++)
	{
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
		#if DIM > 2
			F_coeff[d] = omega[i]*(1-(1/(2*(*tau))))*(((e[d][i]-current_node->u[d])*3)+(e[d][i]*9*((e[0][i]*current_node->u[0])+(e[1][i]*current_node->u[1])+(e[2][i]*current_node->u[2]))));
		#else
			F_coeff[d] = omega[i]*(1-(1/(2*(*tau))))*(((e[d][i]-current_node->u[d])*3)+(e[d][i]*9*((e[0][i]*current_node->u[0])+(e[1][i]*current_node->u[1]))));
		#endif
		}

		force_term = 0;
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
			force_term += F_coeff[d]*current_node->F[d];
		}

		eu = e_mul_u(current_node, e, &i);
		f_eq = current_node->rho*omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	
		current_node->f[i] = current_node->f[i] - (1.0/(*tau)) * (current_node->f[i]-f_eq)+force_term;
	}
}

__device__ __noinline__ void ntpor_collision(Node *current_node, int *opp, int e[DIM][Q], double *omega, double *tau, double *B)
{
	double f_eq, u_sq, eu, collision_bgk, collision_s, tmp[Q];

	u_sq = u_square(current_node);
	for(int i=0;i<Q;i++)
	{
		eu = e_mul_u(current_node, e, &i);
		f_eq = current_node->rho*omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	
		collision_bgk = (1.0/(*tau)) * (current_node->f[i]-f_eq);
		collision_s = current_node->f[opp[i]]-current_node->f[i];
		
		tmp[i] = current_node->f[i] - (1-(*B))*collision_bgk + (*B)*collision_s;
	}

	for(int i =0;i<Q;i++)
	{
		current_node->f[i] = tmp[i];
	}

}

__device__ void guo_ntpor_collision(Node *current_node, int *opp, int e[DIM][Q], double *omega, double *tau, double *B)
{
	double f_eq, u_sq, eu, collision_bgk, collision_s, F_coeff[DIM], force_term, tmp[Q];
	int d;

	#pragma unroll
	for(d = 0; d<DIM; d++)
	{
		current_node->u[d] = current_node->u[d] + (1/2)*current_node->rho*current_node->F[d];
	}

	u_sq = u_square(current_node);

	for(int i=0;i<Q;i++)
	{
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
		#if DIM > 2
			F_coeff[d] = omega[i]*(1-(1/(2*(*tau))))*(((e[d][i]-current_node->u[d])*3)+(e[d][i]*9*((e[0][i]*current_node->u[0])+(e[1][i]*current_node->u[1])+(e[2][i]*current_node->u[2]))));
		#else
			F_coeff[d] = omega[i]*(1-(1/(2*(*tau))))*(((e[d][i]-current_node->u[d])*3)+(e[d][i]*9*((e[0][i]*current_node->u[0])+(e[1][i]*current_node->u[1]))));
		#endif
		}

		force_term = 0;
		#pragma unroll
		for(d = 0; d<DIM; d++)
		{
			force_term += F_coeff[d]*current_node->F[d];
		}

		eu = e_mul_u(current_node, e, &i);
		f_eq = current_node->rho*omega[i]*(1.0+eu+(0.5*eu*eu)-u_sq);
	
		collision_bgk = (1.0/(*tau)) * (current_node->f[i]-f_eq);
		collision_s = current_node->f[opp[i]]-current_node->f[i];

		tmp[i] = current_node->f[i] - (1-(*B))*(collision_bgk) + (*B)*collision_s + (1-(*B))*force_term;
	}

	for(int i =0;i<Q;i++)
	{
		current_node->f[i] = tmp[i];
	}
}

__device__ void bounceback(Node *current_node, int *opp, int e[DIM][Q], double *omega, double *tau, double *B)
{
	double tmp[Q];
	for(int i=0;i<Q;i++)
	{
		tmp[i] = current_node->f[i];
	}

	for(int i=0;i<Q;i++)
	{
		current_node->f[i] = tmp[opp[i]];
	}

	current_node->u[0] = 0;
	current_node->u[1] = 0;
	current_node->rho = 0;
}

#endif
