#include "hip/hip_runtime.h"
#ifndef KERNEL
#define KERNEL
////////////////////////////////////////////////////////////////////////////////
//
// LBM-C
// A lattice Boltzmann fluid flow solver written using CUDA
//
// Copyright (C) 2011  Bruce Jones
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
//
////////////////////////////////////////////////////////////////////////////////
//
// D2Q9 Lattice configuration:
//
//       6   2   5
//        \  |  /
//         \ | /
//          \|/
//       3---0---1
//          /|\
//         / | \
//        /  |  \
//       7   4   8
//
///////////////////////////////////////////////////////////////////////////////

#ifdef _WIN64
	#pragma comment(lib, "cgns/x64/lib/cgns.lib")
	#include "cgns\x64\include\cgnslib.h"
	#pragma comment(lib, "HDF5/x64/lib/hdf5.lib")
	#include "HDF5/x64/include/hdf5.h"
	#pragma comment(lib, "HDF5/x64/lib/libszip.lib")
	#include "HDF5/x64/include/szlib.h"
	#pragma comment(lib, "HDF5/x64/lib/libzlib.lib")
	#include "HDF5/x64/include/zlib.h"
#else
	#pragma comment(lib, "cgns/x86/lib/cgns.lib")
	#include "cgns\x86\include\cgnslib.h"
	#pragma comment(lib, "HDF5/x86/lib/hdf5.lib")
	#include "HDF5/x86/include/hdf5.h"
	#pragma comment(lib, "HDF5/x86/lib/libszip.lib")
	#include "HDF5/x86/include/szlib.h"
	#pragma comment(lib, "HDF5/x86/lib/libzlib.lib")
	#include "HDF5/x86/include/zlib.h"
#endif

#include <stdio.h>
#include "data_types.cuh"
#include "macros.cu"
#include "solver.cu"
#include "index.cuh"
#include "model_builder.cu"
#include "cgns/cgns_output_handler.cu"
#include "cuda_util.cu"

// Include THRUST libraries
#include <thrust/transform_reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/device_vector.h>

// DEVICE VARIABLE DECLARATION
Lattice *lattice_device;
Domain *domain_device;
DomainConstant *domain_constants_device;
OutputController *output_controller_device;

// HOST VARIABLE DECLARATION
Lattice *lattice_host, *lattice_device_prototype;
Domain *domain_host;
DomainConstant *domain_constants_host;
OutputController *output_controller_host;
Timing *times;
ProjectStrings *project;
ModelBuilder model_builder;


// SCALAR DECLARATION (PLATFORM AGNOSTIC)
bool store_macros = false;

// DECLARE OUTPUT HANDLER
CGNSOutputHandler output_handler;

int main(int argc, char **argv)
{

	// Get available memory on graphics card before allocation
	size_t freeMemory_before = 0;
	size_t totalMemory_before = 0;
	hipMemGetInfo(&freeMemory_before, &totalMemory_before);
	
	// Initialise memory for LBM model
	setup(argv[1]);
	
	// Get available memory on graphics card after allocation
	size_t freeMemory_after = 0;
	size_t totalMemory_after = 0;
	hipMemGetInfo(&freeMemory_after, &totalMemory_after);

	// Report program memory usage
	cout << "Total Device Memory:	 "<< totalMemory_after / 1024 / 1024 << "Mb" << endl;
	cout << "Total Availabe Memory:	 "<< freeMemory_before / 1024 / 1024 << "Mb" << endl;
	cout << "Memory Used:            "<< (freeMemory_before-freeMemory_after) / 1024 / 1024 << "Mb" << endl;

	// Report domain configuration
	printf("X-Length:		%d\n", domain_constants_host->length[0]);
	printf("Y-Length:		%d\n", domain_constants_host->length[1]);
	#if DIM > 2
		printf("Z-Length:		%d\n", domain_constants_host->length[2]);
	#endif
	printf("Relaxation Time (Tau):	%f\n", domain_constants_host->tau);
	printf("\nPress return to continue...");
	if (output_controller_host->interactive == true) getchar();

	domain_constants_host->residual = 0;

	// Get current clock cycle number
	clock_t t1=clock();

	int domain_size=1;
	for(int d = 0; d<DIM ;d++)
	{
		domain_size = domain_size*domain_constants_host->length[d];
	}

	int coord[DIM];
	coord[0] = floor((float)domain_constants_host->length[0]/2.);
	//coord[0] = 0;
	coord[1] = floor((float)domain_constants_host->length[1]/2.);
	//coord[1] = 0;
	#if DIM > 2
		//coord[2] = floor((float)domain_constants_host->length[2]/2.);
		coord[2] = 0;
	#endif
	for(int i = 1; i<times->max+1; i++)
	{
		if((times->plot>0 && i%times->plot == 0) ||
		   (times->steady_check>0 && i%times->steady_check) || 
		   (times->screen>0 && i%times->screen)) store_macros = true;

		iterate();

		if(times->plot>0 && i%times->plot == 0)
		{
			output_macros(i);
			store_macros = false;
		}

		if(times->screen>0 && i%times->screen == 0)
		{
			screen_mess(i,coord);
			store_macros = false;
		}

		if(times->steady_check>0 && i%times->steady_check == 0)
		{
			compute_residual();
			if(isIndeterminate(domain_constants_host->residual))
			{
				output_macros(i);
				exit(1);
			} else if(domain_constants_host->residual<domain_constants_host->tolerance)
			{
				output_macros(i);
				break;
			}
			store_macros = false;
		}
	}

	// Get current clock cycle number
	clock_t t2=clock();
	// Compare and report global execution time
	double cputime = ((double)t2-(double)t1)/(double)CLOCKS_PER_SEC;
	printf("\n\nTotal Run Time: %fs",cputime);
	printf("\nPress return to finish");
	if (output_controller_host->interactive == true) getchar();


}


// EXECUTES ALL ROUTINES REQUIRED FOR THE MODEL SET UP
void setup(char *data_file)
{
	// Set cuda device to use
	hipSetDevice(0);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(iterate_kernel), hipFuncCachePreferL1);
	
	// Allocate container structures
	combi_malloc<Lattice>(&lattice_host, &lattice_device, sizeof(Lattice));
	combi_malloc<Domain>(&domain_host, &domain_device, sizeof(Domain));
	combi_malloc<DomainConstant>(&domain_constants_host, &domain_constants_device, sizeof(DomainConstant));
	combi_malloc<OutputController>(&output_controller_host, &output_controller_device, sizeof(OutputController));
	domain_constants_host = (DomainConstant *)malloc(sizeof(DomainConstant));
	times = (Timing *)malloc(sizeof(Timing));
	project = (ProjectStrings *)malloc(sizeof(ProjectStrings));
	lattice_device_prototype = (Lattice *)malloc(sizeof(Lattice));

	ModelBuilder tmpmb(data_file, lattice_host, lattice_device,
		domain_constants_host, domain_constants_device,
		domain_host, domain_device,
		output_controller_host, output_controller_device,
		times, project);
	model_builder = tmpmb;

	/*model_builder.get_model(lattice_host, lattice_device,
		domain_constants_host, domain_constants_device,
		domain_arrays_host, domain_arrays_device,
		output_controller_host, output_controller_device,
		times, project);*/
	int z_len = 1;
	#if DIM > 2
		z_len = domain_constants_host->length[2];
	#endif
	CGNSOutputHandler tmp(project->output_fname,domain_constants_host->length[0],domain_constants_host->length[1],z_len);
	output_handler = tmp;
}



// COPIES f_i DATA FROM DEVICE TO HOST AND COMPUTERS MACROSCOPIC VALUES ON HOST, THIS DATA
// IS THEN WRITTEN TO THE OUTPUT FILE
//
// Note:	A computationally more efficient implementation would compute macroscopic
//			value's on the gpu and then just copy that data, this would however consume
//			more memory
void output_macros(int time)
{
	int domain_size = domain_constants_host->length[0]*domain_constants_host->length[1];
	#if DIM > 2
		domain_size = domain_size*domain_constants_host->length[2];
	#endif

	Domain domain_tmp;

	cudasafe(hipMemcpy(&domain_tmp, domain_device, sizeof(Domain),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");
	
	double *u_tmp[DIM];
	cudasafe(hipMemcpy(u_tmp, domain_tmp.u, sizeof(double*)*DIM,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	for(int d=0;d<DIM;d++)
	{
		cudasafe(hipMemcpy(domain_host->u[d], u_tmp[d], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");
	}

	double *rho_tmp;
	cudasafe(hipMemcpy(domain_host->rho, domain_tmp.rho, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	// Copy data from device to host
	//cudasafe(hipMemcpy(lattice_host->rho, lattice_device->rho, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - rho");
	//cudasafe(hipMemcpy(lattice_host->u[0], lattice_device->u[0], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - ux");
	//cudasafe(hipMemcpy(lattice_host->u[1], lattice_device->u[1], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - uy");

	int num_fields = 0;
	if (output_controller_host->u[0] == true) num_fields++;
	if (output_controller_host->u[1] == true) num_fields++;
#if DIM > 2
	if (output_controller_host->u[2] == true) num_fields++;
#endif
	if (output_controller_host->rho == true) num_fields++;

	char **labels;
	double **data;

	labels = (char **)malloc(num_fields * sizeof (char *));
	data = (double **)malloc(num_fields * sizeof(double));

	for(int i = 0; i<num_fields;i++)
	{
		labels[i] = (char *)malloc(STR_LENGTH*sizeof(char));
	}

	int counter = 0;

	if (output_controller_host->u[0] == true)
	{
		data[counter] = domain_host->u[0];
		strcpy(labels[counter],"VelocityX");
		counter++;
	}

	if (output_controller_host->u[1] == true)
	{
		data[counter] = domain_host->u[1];
		strcpy(labels[counter],"VelocityY");
		counter++;
	}
#if DIM > 2
	if (output_controller_host->u[2] == true)
	{
		data[counter] = domain_host->u[2];
		strcpy(labels[counter],"VelocityZ");
		counter++;
	}
#endif	
	if (output_controller_host->rho == true)
	{
		data[counter] = domain_host->rho;
		strcpy(labels[counter],"Density");
		counter++;
	}

/*	data[0] = lattice_host->rho;
	data[1] = lattice_host->u[0];
	data[2] = lattice_host->u[1];

	strcpy(labels[0],"Density");
	strcpy(labels[1],"VelocityX");
	strcpy(labels[2],"VelocityY");*/

	output_handler.append_solution_output(time,num_fields,data,labels);
}

// CONFIGURES THE KERNEL CONFIGURATION AND LAUNCHES KERNEL
void iterate(void)
{
	// GRID AND BLOCK DEFINITIONS CAN BE CALCULATED BEFORE ITERATE
	// DEFINE GRID AND BLOCK DIMS
	int3 threads;
	threads.x = (int)ceilf((float)domain_constants_host->length[0]/(float)NUM_THREADS_DIM_X);
	threads.y = (int)ceilf((float)domain_constants_host->length[1]/(float)NUM_THREADS_DIM_Y);
	threads.z = 1;

	int3 blocks;
	blocks.x = NUM_THREADS_DIM_X;
	blocks.y = NUM_THREADS_DIM_Y;
	blocks.z = 1;

	#if DIM >2
		threads.z = (int)ceilf((float)domain_constants_host->length[2]/(float)NUM_THREADS_DIM_Z);;
		blocks.z = NUM_THREADS_DIM_Z;
	#endif

	dim3 grid_dim = dim3(threads.x,threads.y,threads.z);
    dim3 block_dim = dim3(blocks.x,blocks.y,blocks.z);
	hipDeviceSynchronize();
	Check_CUDA_Error("Kernel \"iterate_bulk 1\" Execution Failed!");  
	// ITERATE ONCE
	iterate_kernel<<<grid_dim, block_dim>>>(lattice_device, domain_device, store_macros);
	hipDeviceSynchronize();
	Check_CUDA_Error("Kernel \"iterate_bulk 1\" Execution Failed!");  
	// SWAP CURR AND PREV LATTICE POINTERS READY FOR NEXT ITER
	swap_lattices();
}

void swap_lattices(void)
{
	cudasafe(hipMemcpy(lattice_device_prototype, lattice_device, sizeof(Lattice),hipMemcpyDeviceToHost),"Copy Data: Device Lattice Pointers From Device");
	double **tmp_1 = lattice_device_prototype->f_prev;
	double **tmp_2 = lattice_device_prototype->f_curr;
	lattice_device_prototype->f_curr = tmp_1;
	lattice_device_prototype->f_prev = tmp_2;
	cudasafe(hipMemcpy(lattice_device, lattice_device_prototype, sizeof(Lattice),hipMemcpyHostToDevice),"Copy Data: Device Lattice Pointers To Device");
}

// square<T> computes the square of a number f(x) -> x*x
/*template <typename T>
struct square
{
    __host__ __device__
        T operator()(const T& x) const { 
            return x * x;
        }
};

template <typename T>
struct velocity
{
    __host__ __device__
        T operator()(const T& x, const T& y, const T& z) const { 
            return (x * x) + (y * y) + (z * z);
        }
};*/

/*template <typename T>
struct total_energy
{
    __host__ __device__
        T operator()(const T& x, const T& y, const T& z, const T& rho) const { 
            return 0.5*rho*((x * x) + (y * y) + (z * z));
        }
};*/

struct energy
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        thrust::get<4>(t) = 0.5*thrust::get<3>(t)*((thrust::get<0>(t)*thrust::get<0>(t)) + (thrust::get<1>(t)*thrust::get<1>(t)) + (thrust::get<2>(t)*thrust::get<2>(t)));
    }
};

double current_RMS(double *device_var_x, double *device_var_y, double *device_var_z, double *device_var_rho, int var_size)
{
	double *result;
	cudasafe(hipMalloc((void **)&result,sizeof(double)*var_size), "Model Builder: Device memory allocation failed!");

	// wrap raw pointer with a device_ptr for thrust compatibility
	thrust::device_ptr<double> dev_ptr_x(device_var_x);
	thrust::device_ptr<double> dev_ptr_y(device_var_y);
	thrust::device_ptr<double> dev_ptr_z(device_var_z);
	thrust::device_ptr<double> dev_ptr_rho(device_var_rho);
	thrust::device_ptr<double> dev_ptr_res(result);

	// apply the transformation
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(dev_ptr_x, dev_ptr_y, dev_ptr_z, dev_ptr_rho, dev_ptr_res)),
                     thrust::make_zip_iterator(thrust::make_tuple(dev_ptr_x+var_size, dev_ptr_y+var_size, dev_ptr_z+var_size, dev_ptr_rho+var_size, dev_ptr_res+var_size)),
                     energy());
	Check_CUDA_Error("Kernel \"iterate_bulk 1\" Execution Failed!");  
    
	double init = 0;

	// Compute RMS value
	double sum = thrust::reduce(dev_ptr_res, dev_ptr_res+var_size, (double) 0, thrust::plus<double>());

	double curr_RMS = sqrt(sum/var_size);
	cout << "RESIDUAL = " << curr_RMS << endl;
	cout << "SUM = " << sum << endl;
	cout << "VAR_SIZE = " << var_size << endl;

	return curr_RMS;
}

double prev_RMS = 0;

double error_RMS(double *device_var_x, double *device_var_y, double *device_var_z, double *device_var_rho, int var_size)
{
	double curr_RMS = current_RMS(device_var_x, device_var_y, device_var_z, device_var_rho, var_size);
	double tmp = abs(curr_RMS-prev_RMS);

	prev_RMS = curr_RMS;

	return tmp;
}

void compute_residual(void)
{
	int domain_size = domain_constants_host->length[0]*domain_constants_host->length[1];
	#if DIM > 2
		domain_size = domain_size*domain_constants_host->length[2];
	#endif

	Domain domain_tmp;

	cudasafe(hipMemcpy(&domain_tmp, domain_device, sizeof(Domain),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	double *u_tmp[DIM];
	cudasafe(hipMemcpy(u_tmp, domain_tmp.u, sizeof(double*)*DIM,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	//double *rho_tmp;
	//cudasafe(hipMemcpy(rho_tmp, domain_tmp.rho, sizeof(double*),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	/*cudasafe(hipMemcpy(domain_host->u[0], u_tmp[0], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - u");
	cudasafe(hipMemcpy(domain_host->u[1], u_tmp[1], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - u");
	cudasafe(hipMemcpy(domain_host->u[2], u_tmp[2], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - u");*/

//	domain_constants_host->residual = error_RMS(u_tmp[0],u_tmp[1],u_tmp[2], rho_tmp,domain_size);
	domain_constants_host->residual = error_RMS(u_tmp[0],u_tmp[1],u_tmp[2], domain_tmp.rho,domain_size);
}

void screen_mess(int iter, int coord[DIM])
{
	int idx = coord[0]+coord[1]*domain_constants_host->length[0];
	#if DIM > 2
		idx += coord[2]*domain_constants_host->length[0]*domain_constants_host->length[1];
	#endif

	double u[DIM],rho;
	Domain domain_tmp;

	cudasafe(hipMemcpy(&domain_tmp, domain_device, sizeof(Domain),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");
	
	double *u_tmp[DIM];
	cudasafe(hipMemcpy(u_tmp, domain_tmp.u, sizeof(double*)*DIM,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	for(int d=0;d<DIM;d++)
	{
		cudasafe(hipMemcpy(&u[d], &u_tmp[d][idx], sizeof(double),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");
	}

	cudasafe(hipMemcpy(&rho, &domain_tmp.rho[idx], sizeof(double),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	cout << "time = " << iter << "; rho = " << rho << "; uX = " << u[0]<< "; uY = " << u[1] << "; ";
	#if DIM>2
		cout << "uZ = " << u[2] << "; ";
	#endif
	cout << "resid = " << domain_constants_host->residual << endl;
}

bool isIndeterminate(const double pV)
{
    return (pV != pV);
} 

#endif