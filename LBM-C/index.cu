#include "hip/hip_runtime.h"
#ifndef KERNEL
#define KERNEL
////////////////////////////////////////////////////////////////////////////////
//
// LBM-C
// A lattice Boltzmann fluid flow solver written using CUDA
//
// Copyright (C) 2011  Bruce Jones
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
//
////////////////////////////////////////////////////////////////////////////////
//
// D2Q9 Lattice configuration:
//
//       6   2   5
//        \  |  /
//         \ | /
//          \|/
//       3---0---1
//          /|\
//         / | \
//        /  |  \
//       7   4   8
//
///////////////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include "data_types.cuh"
#include "macros.cu"
#include "solver.cu"
#include "index.cuh"

// Include THRUST libraries
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

// DEVICE VARIABLE DECLARATION
Lattice *lattice_device;
Domain *domain_device;
double *f_1_device, *f_2_device, *rho_device, *ux_device, *uy_device, *u_device, *boundary_value_device, *geometry_device; 
int *boundary_type_device;

// HOST VARIABLE DECLARATION
Lattice *lattice_host, *lattice_device_prototype;
Domain *domain_host;
double *f_host, *rho_host, *ux_host, *uy_host, *u_host, *boundary_value_host, *geometry_host;
int *boundary_type_host;

// SCALAR DECLARATION (PLATFORM AGNOSTIC)
double tau, residual;
double tolerance;
int domain_size, l_b_o, maxT, saveT, steadyT;
int3 length;
bool store_macros = false;

int main(int argc, char **argv)
{
	//tolerance = 0.00000001;

	// Get available memory on graphics card before allocation
	size_t freeMemory_before;
	size_t totalMemory_before;
	hipMemGetInfo(&freeMemory_before, &totalMemory_before);
	
	// Initialise memory for LBM model
	setup();

	// Get available memory on graphics card after allocation
	size_t freeMemory_after;
	size_t totalMemory_after;
	hipMemGetInfo(&freeMemory_after, &totalMemory_after);

	// Report program memory usage
	printf("Total Device Memory:	%luMb\n", (unsigned long) totalMemory_after / 1024 / 1024);
	printf("Total Availabe Memory:	%luMb\n", (unsigned long) freeMemory_before / 1024 / 1024);
	printf("Memory Used:		%luMb\n\n", (unsigned long) (freeMemory_before-freeMemory_after) / 1024 / 1024);

	// Report domain configuration
	printf("Length.x:		%d\n", domain_host->length.x);
	printf("Length.y:		%d\n", domain_host->length.y);
	printf("Relaxation Time (Tau):	%f\n", domain_host->tau);
	printf("\nPress return to continue...");
	getchar();

	residual = 0;
	output_macros(-1);

	// Get current clock cycle number
	clock_t t1=clock();

	for(int i = 0; i<maxT; i++)
	{
		//iterate();
		if(i%saveT == 0 && steadyT>0 && i%steadyT)
		{
			store_macros = true;
			iterate();
			output_macros(i);
			residual = error_RMS(u_device,domain_size);
			if(residual<tolerance) break;
			store_macros = false;
		} else if (i%saveT==0)
		{
			store_macros = true;
			iterate();
			output_macros(i);
			store_macros = false;
		} else if(steadyT>0 && i%steadyT)
		{
			store_macros = true;
			iterate();
			cudasafe(hipMemcpy(u_host, u_device, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - u");
			residual = error_RMS(u_device,domain_size);
			if(residual<tolerance) break;
			store_macros = false;
		} else{
			iterate();
		}
	}

	// Get current clock cycle number
	clock_t t2=clock();
	// Compare and report global execution time
	double cputime = ((double)t2-(double)t1)/(double)CLOCKS_PER_SEC;
	printf("\n\nTotal Run Time: %fs",cputime);
	printf("\nPress return to finish");
	getchar();


}

// ALLOCATES MEMORY ON THE HOST
void allocate_memory_host(void)
{
	// ALLOCATE ARRAY AND STRUCT MEMORY ON HOST
	// STRUCTS:
	lattice_host = (Lattice *)malloc(sizeof(Lattice));
	domain_host = (Domain *)malloc(sizeof(Domain));
	// ARRAYS:
	boundary_type_host = (int *)malloc(domain_size*sizeof(int));
	boundary_value_host = (double *)malloc(domain_size*sizeof(double));
	geometry_host = (double *)malloc(domain_size*sizeof(double));
	f_host = (double *)malloc(domain_size*Q*sizeof(double));
	rho_host = (double *)malloc(domain_size*sizeof(double));
	ux_host = (double *)malloc(domain_size*sizeof(double));
	uy_host = (double *)malloc(domain_size*sizeof(double));
	u_host = (double *)malloc(domain_size*sizeof(double));
}

// ALLOCATES MEMORY ON THE DEVICE
void allocate_memory_device(void)
{
	// ALLOCATE ARRAY AND STRUCT MEMORY ON DEVICE
	// STRUCTS:
	cudasafe(hipMalloc((void **)&lattice_device,sizeof(Lattice)), "Allocate Memory: lattice_device");
	cudasafe(hipMalloc((void **)&domain_device,sizeof(Domain)), "Allocate Memory: control_device");
	// ARRAYS:
	cudasafe(hipMalloc((void **)&f_1_device,domain_size*Q*sizeof(double)), "Allocate Memory: f_1_device");
	cudasafe(hipMalloc((void **)&f_2_device,domain_size*Q*sizeof(double)), "Allocate Memory: f_2_device");
	cudasafe(hipMalloc((void **)&rho_device,domain_size*Q*sizeof(double)), "Allocate Memory: rho_device");
	cudasafe(hipMalloc((void **)&ux_device,domain_size*Q*sizeof(double)), "Allocate Memory: ux_device");
	cudasafe(hipMalloc((void **)&uy_device,domain_size*Q*sizeof(double)), "Allocate Memory: uy_device");
	cudasafe(hipMalloc((void **)&u_device,domain_size*Q*sizeof(double)), "Allocate Memory: u_device");
	cudasafe(hipMalloc((void **)&boundary_type_device,domain_size*sizeof(int)), "Allocate Memory: boundary_type_device");
	cudasafe(hipMalloc((void **)&boundary_value_device,domain_size*sizeof(double)), "Allocate Memory: boundary_value_device");
	cudasafe(hipMalloc((void **)&geometry_device,domain_size*sizeof(double)), "Allocate Memory: geometry_device");

}

// READS INPUT DATA FROM FILE AND ASSEMBLES DATA INTO RELEVANT STRUCTS
void load_and_assemble_data(void)
{
	// ASSEMBLE STRUCT ON HOST: Lattice
	lattice_host->f_curr = f_host;
	lattice_host->rho = rho_host;
	lattice_host->ux = ux_host;
	lattice_host->uy = uy_host;
	lattice_host->u = u_host;

	// ASSEMBLE AND LOAD STRUCT ON HOST: Control
	// ASSEMBLE
	domain_host->boundary_type = boundary_type_host;
	domain_host->boundary_value = boundary_value_host;
	domain_host->geometry = geometry_host;
	// LOAD
	domain_host->tau = tau;
	domain_host->length.x = length.x;
	domain_host->length.y = length.y;
	
	// Boundary nodes are treated as chains of face nodes, vertex nodes and corner nodes,
	// the length of each of these chains is a function of domain dimensions and is calculated
	// here.
	domain_host->b_o[0] = (length.y-2); // X-
	domain_host->b_o[1] = domain_host->b_o[0]+(length.y-2); // X+
	domain_host->b_o[2] = domain_host->b_o[1]+(length.x-2); // Y-
	domain_host->b_o[3] = domain_host->b_o[2]+(length.x-2); // Y+
	domain_host->b_o[4] = domain_host->b_o[3]+4;
	l_b_o = domain_host->b_o[4];

	// ASSEMBLE STRUCT ON DEVICE: Lattice
	lattice_device_prototype = (Lattice *)malloc(sizeof(Lattice));
	lattice_device_prototype->f_curr = f_1_device;
	lattice_device_prototype->f_prev = f_2_device;
	lattice_device_prototype->rho = rho_device;
	lattice_device_prototype->ux = ux_device;
	lattice_device_prototype->uy = uy_device;
	lattice_device_prototype->u = u_device;
	cudasafe(hipMemcpy(lattice_device, lattice_device_prototype, sizeof(Lattice),hipMemcpyHostToDevice),"Copy Data: lattice_device");

	// ASSEMBLE AND LOAD STRUCT ON DEVICE: Control
	Domain *domain_tmp = (Domain *)malloc(sizeof(Domain));
	domain_tmp->tau = tau;
	domain_tmp->length.x = length.x;
	domain_tmp->length.y = length.y;
	domain_tmp->boundary_type = boundary_type_device;
	domain_tmp->boundary_value = boundary_value_device;
	domain_tmp->geometry = geometry_device;
	cudasafe(hipMemcpy(domain_device, domain_tmp, sizeof(Domain),hipMemcpyHostToDevice),"Copy Data: control_device");
	cudasafe(hipMemcpy(&domain_device->b_o, &domain_host->b_o, sizeof(int)*5,hipMemcpyHostToDevice),"Copy Data: b_o");
}

// CALCULATES AND LOADS A CONSTANT DENSITY ZERO VELOCITY INITIAL CONDITION FOR THE DOMAIN
void load_static_IC(void)
{
	int index_i;
	double omega[Q];
	LOAD_OMEGA(omega);
	for(int i=0;i<Q;i++)
	{
		for(int index=0;index<(domain_size);index++)
		{
			index_i = index+i*(domain_size);
			lattice_host->f_curr[index_i] = 1.f*omega[i];
		}
	}
	cudasafe(hipMemcpy(f_2_device, f_host, sizeof(double)*Q*domain_size,hipMemcpyHostToDevice),"Copy Data: Initial Condition");
}

// EXECUTES ALL ROUTINES REQUIRED FOR THE MODEL SET UP
void setup(void)
{
	// Set cuda device to use
	hipSetDevice(0);

	// Read domain configuration
	FILE * input_file;
    input_file = fopen ("input.dat","r");
	int IC_type, i2d;
	//IC_type = 0;
	fscanf(input_file,"%d %d %lf %d %d %d %lf %d\n", &length.x, &length.y, &tau, &saveT, &maxT, &steadyT, &tolerance, &IC_type);
	domain_size = length.x*length.y;
	allocate_memory_host();
	allocate_memory_device();
	load_and_assemble_data();
	if (IC_type == 0) load_static_IC();
	for(int j = 0; j<length.y; j++)
	{
		for(int i = 0; i<length.x; i++)
		{
			i2d = i + j*length.x;
			fscanf(input_file,"%d %lf\n", &domain_host->boundary_type[i2d], &domain_host->boundary_value[i2d]);
		}
	}

	for(int j = 0; j<length.y; j++)
	{
		for(int i = 0; i<length.x; i++)
		{
			i2d = i + j*length.x;
			fscanf(input_file,"%f\n", &domain_host->geometry[i2d]);
		}
	}

	cudasafe(hipMemcpy(boundary_type_device, boundary_type_host, sizeof(int)*domain_size,hipMemcpyHostToDevice),"Copy Data: boundary_type_device");
	cudasafe(hipMemcpy(boundary_value_device, boundary_value_host, sizeof(double)*domain_size,hipMemcpyHostToDevice),"Copy Data: boundary_value_device");
	cudasafe(hipMemcpy(geometry_device, geometry_host, sizeof(double)*domain_size,hipMemcpyHostToDevice),"Copy Data: geometry_device");

}

// ERROR CHECKING FOR MEMORY ALLOCATION
void cudasafe( hipError_t error, char* message)
{
   if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}

// ERROR CHECKING FOR KERNEL EXECUTION
void Check_CUDA_Error(const char *message)
{
   hipError_t error = hipGetLastError();
   if(error!=hipSuccess) {
      fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
      exit(-1);
   }                         
}

// COPIES f_i DATA FROM DEVICE TO HOST AND COMPUTERS MACROSCOPIC VALUES ON HOST, THIS DATA
// IS THEN WRITTEN TO THE OUTPUT FILE
//
// Note:	A computationally more efficient implementation would compute macroscopic
//			value's on the gpu and then just copy that data, this would however consume
//			more memory
void output_macros(int time)
{
	// Copy data from device to host
	cudasafe(hipMemcpy(rho_host, rho_device, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - rho");
	cudasafe(hipMemcpy(ux_host, ux_device, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - ux");
	cudasafe(hipMemcpy(uy_host, uy_device, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - uy");
	cudasafe(hipMemcpy(u_host, u_device, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - u");
	
// Assemble formatted filename	
	char fname[19];
	FILE *file;
	sprintf(fname, "results_%i.dat", time);
// Open file
	file = fopen(fname,"w");
// Write File Header	
	fprintf(file,"TITLE=\"2D Poiseuille Flow\"\nVARIABLES= \"X\", \"Y\", \"rho\", \"uX\", \"uY\", \"u\"");//\nDATASETAUXDATA ComputerTime=\"%lus\"\nDATASETAUXDATA DeviceMemoryUsed=\"%luMb\"",cputime, mem);
// Write Zone Header
	// note: nx and ny values are not in the "correct" order in the zone header, errors occur when loading the data in tecplot
	// if the "correct" order is used
	fprintf(file,"\nZONE T=\"2D Poiseuille Flow at time = %i\", I=%i, J=%i, DATAPACKING=POINT, SOLUTIONTIME=%i", time,length.x,length.y,time);
// Loop over all nodes to calculate and print nodal macroscopic values to file, output some feedback data to console
	int i2d;
	for (int y=0;y<length.y;y++){
		for (int x=0;x<length.x;x++){
			
			i2d = x+y*length.x;

			// Impose zero velocity on bounceback nodes
			if(domain_host->geometry[i2d] == 1)
			{
				lattice_host->ux[i2d] = 0;
				lattice_host->uy[i2d] = 0;
				lattice_host->u[i2d] = 0;
			}
			// Write to files
			fprintf(file,"\n%i %i %lf %lf %lf %lf", x, y, lattice_host->rho[i2d], lattice_host->ux[i2d], lattice_host->uy[i2d], lattice_host->u[i2d]);
			// Output reference information to console
			if (y==length.y/2 && x == 0) {printf("\n time = %i; rho = %lf; uX = %lf; uY = %lf, resid = %g", time, lattice_host->rho[i2d], lattice_host->ux[i2d], lattice_host->uy[i2d], residual);}
			//if (y==length.y/2 && x == 0) {printf("\n time = %i; resid = %e", time, residual);}
		}
	}
	// Close file
	fclose(file);
}

// CONFIGURES THE KERNEL CONFIGURATION AND LAUNCHES KERNEL A KERNEL BOTH FOR THE BOUNDARY NODES
// AND THE BULK NODES
//
// Note:	The "all" kernel operates cleanly on both bulk and boundary nodes and may be used
//			instead, though its use is inefficient.
void iterate(void)
{
	// GRID AND BLOCK DEFINITIONS CAN BE CALCULATED BEFORE ITERATE
	// DEFINE BULK GRID AND BLOCK
	dim3 Db_bulk = dim3(length.x-2,1,1);
    dim3 Dg_bulk = dim3(length.y-2,1,1);
	// DEFINE BOUNDARY GRID AND BLOCK
	int boundary_amount = l_b_o;
	int boundary_grid=(int)(boundary_amount/BLOCK_SIZE);
	int boundary_leftover=(boundary_amount%BLOCK_SIZE);

	// ITERATE ONCE
	iterate_bulk_kernel<<<Dg_bulk, Db_bulk>>>(lattice_device, domain_device, store_macros);
	Check_CUDA_Error("Kernel \"iterate_bulk 1\" Execution Failed!");  
	iterate_boundary_kernel<<<boundary_grid,BLOCK_SIZE>>>(lattice_device,domain_device,0, store_macros);
	Check_CUDA_Error("Kernel \"iterate_boundary 1a\" Execution Failed!");  
	if(boundary_leftover)
		iterate_boundary_kernel<<<1,boundary_leftover>>>(lattice_device,domain_device,boundary_amount-boundary_leftover, store_macros);
	Check_CUDA_Error("Kernel \"iterate_boundary 1b\" Execution Failed!");

	// SWAP CURR AND PREV LATTICE POINTERS READY FOR NEXT ITER
	cudasafe(hipMemcpy(lattice_device_prototype, lattice_device, sizeof(Lattice),hipMemcpyDeviceToHost),"Copy Data: Device Lattice Pointers");
	double *tmp_1 = lattice_device_prototype->f_prev;
	double *tmp_2 = lattice_device_prototype->f_curr;
	lattice_device_prototype->f_curr = tmp_1;
	lattice_device_prototype->f_prev = tmp_2;
	cudasafe(hipMemcpy(lattice_device, lattice_device_prototype, sizeof(Lattice),hipMemcpyHostToDevice),"Copy Data: Device Lattice Pointers");
}
/*
void iterate(void)
{
	// GRID AND BLOCK DEFINITIONS CAN BE CALCULATED BEFORE ITERATE
	// DEFINE all GRID AND BLOCK
	dim3 Db_all = dim3(length.x-2,1,1);
    dim3 Dg_all = dim3(length.y-2,length.z-2,1);
	// DEFINE all GRID AND BLOCK
	int all_amount = l_b_o;
	int all_grid=(int)(all_amount/BLOCK_SIZE);
	int all_leftover=(all_amount%BLOCK_SIZE);

	// ITERATE ONCE
	iterate_all_kernel<<<Dg_all, Db_all>>>(lattice_1_device,lattice_2_device,domain_device,0,1);
	Check_CUDA_Error("Kernel \"iterate_all 1\" Execution Failed!");  
	iterate_all_kernel<<<all_grid,BLOCK_SIZE>>>(lattice_1_device,lattice_2_device,domain_device,0,0);
	Check_CUDA_Error("Kernel \"iterate_all 1a\" Execution Failed!");  
	if(all_leftover)
		iterate_all_kernel<<<1,all_leftover>>>(lattice_1_device,lattice_2_device,domain_device,all_amount-all_leftover,0);
	Check_CUDA_Error("Kernel \"iterate_all 1b\" Execution Failed!");

	// SWAP LATTICES AND ITERATE AGAIN
	iterate_all_kernel<<<Dg_all, Db_all>>>(lattice_2_device,lattice_1_device,domain_device,0,1);
	Check_CUDA_Error("Kernel \"iterate_all 2\" Execution Failed!");  
	iterate_all_kernel<<<all_grid,BLOCK_SIZE>>>(lattice_2_device,lattice_1_device,domain_device,0,0);
	Check_CUDA_Error("Kernel \"iterate_all 1a\" Execution Failed!");  
	if(all_leftover)
		iterate_all_kernel<<<1,all_leftover>>>(lattice_2_device,lattice_1_device,domain_device,all_amount-all_leftover,0);
	Check_CUDA_Error("Kernel \"iterate_all 1b\" Execution Failed!");  
}*/

// square<T> computes the square of a number f(x) -> x*x

template <typename T>
struct square
{
    __host__ __device__
        T operator()(const T& x) const { 
            return x * x;
        }
};

double current_RMS(double *device_var, int var_size)
{
	// wrap raw pointer with a device_ptr for thrust compatibility
	thrust::device_ptr<double> dev_ptr(device_var);

	// setup arguments for thrust transformation to square array elements then execute plus reduction
    square<double>        unary_op;
    thrust::plus<double> binary_op;
    double init = 0;

	// Compute RMS value
	double sum = thrust::transform_reduce(dev_ptr, dev_ptr+var_size, unary_op, init, binary_op);

	double curr_RMS = sqrt(sum/var_size);

	return curr_RMS;
}

double prev_RMS = 0;

double error_RMS(double *device_var, int var_size)
{
	double curr_RMS = current_RMS(device_var, var_size);
	double tmp = abs(curr_RMS-prev_RMS);

	prev_RMS = curr_RMS;

	return tmp;
}

#endif