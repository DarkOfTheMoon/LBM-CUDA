#include "hip/hip_runtime.h"
#ifndef KERNEL
#define KERNEL
////////////////////////////////////////////////////////////////////////////////
//
// This is a 15 velocity set method:
// Distribution functions are stored as "f" arrays
// Think of these as the number of particles moving in these directions:
//
//      f6  f2   f5
//        \  |  /
//         \ | /
//          \|/
//      f3---|--- f1
//          /|\
//         / | \       and f0 for the rest (zero) velocity
//        /  |  \
//      f7  f4   f8
//
///////////////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "data_types.cuh"
#include "macros.cu"
#include "d2q9_boundary.cu"
#include "solver.cuh"
#include "index.cuh"

// DEVICE VARIABLE DECLARATION
Lattice *lattice_1_device, *lattice_2_device;
Domain *domain_device;
float *f_1_device, *f_2_device, *boundary_value_device, *boundary_type_device; 

// HOST VARIABLE DECLARATION
Lattice *lattice_host;
Domain *domain_host;
Output *output;
float *f_host, *rho, *ux, *uy, *uz, *u, *boundary_value_host, *boundary_type_host;

// SCALAR DECLARATION (PLATFORM AGNOSTIC)
float tau;
int domain_size, l_b_o, maxT, saveT;
int3 length;

int main(int argc, char **argv)
{
	// Get available memory on graphics card before allocation
	size_t freeMemory_before;
	size_t totalMemory_before;
	hipMemGetInfo(&freeMemory_before, &totalMemory_before);
	
	// Initialise memory for LBM model
	setup();

	// Get available memory on graphics card after allocation
	size_t freeMemory_after;
	size_t totalMemory_after;
	hipMemGetInfo(&freeMemory_after, &totalMemory_after);

	// Report program memory usage
	printf("Total Device Memory:	%luMb\n", (unsigned long) totalMemory_after / 1024 / 1024);
	printf("Total Availabe Memory:	%luMb\n", (unsigned long) freeMemory_before / 1024 / 1024);
	printf("Memory Used:		%luMb\n\n", (unsigned long) (freeMemory_before-freeMemory_after) / 1024 / 1024);

	// Report domain configuration
	printf("Length.x:		%d\n", domain_host->length.x);
	printf("Length.y:		%d\n", domain_host->length.y);
	printf("Length.z:		%d\n", domain_host->length.z);
	printf("Relaxation Time (Tau):	%f\n", domain_host->tau);
	printf("\nPress the any key to continue...");
	getchar();

	output_macros(-1);

	// Get current clock cycle number
	clock_t t1=clock();

	for(int i = 0; i<(maxT/2); i++)
	{
		iterate();
		if((2*i)%(saveT) == 0)
		{
			output_macros(2*i);
		}
	}
	
	// Get current clock cycle number
	clock_t t2=clock();
	// Compare and report global execution time
	double cputime = ((double)t2-(double)t1)/(double)CLOCKS_PER_SEC;
	printf("\n\nTotal Run Time: %fs",cputime);
	printf("\nPress the any key to finish");
	getchar();


}

// ALLOCATES MEMORY ON THE HOST
void allocate_memory_host(void)
{
	// ALLOCATE ARRAY AND STRUCT MEMORY ON HOST
	// STRUCTS:
	lattice_host = (Lattice *)malloc(sizeof(Lattice));
	domain_host = (Domain *)malloc(sizeof(Domain));
	output = (Output *)malloc(sizeof(Output));
	// ARRAYS:
	boundary_type_host = (float *)malloc(domain_size*sizeof(float));
	boundary_value_host = (float *)malloc(domain_size*sizeof(float));
	f_host = (float *)malloc(domain_size*Q*sizeof(float));
	rho = (float *)malloc(domain_size*sizeof(float));
	ux = (float *)malloc(domain_size*sizeof(float));
	uy = (float *)malloc(domain_size*sizeof(float));
	uz = (float *)malloc(domain_size*sizeof(float));
	u = (float *)malloc(domain_size*sizeof(float));
}

// ALLOCATES MEMORY ON THE DEVICE
void allocate_memory_device(void)
{
	// ALLOCATE ARRAY AND STRUCT MEMORY ON DEVICE
	// STRUCTS:
	cudasafe(hipMalloc((void **)&lattice_1_device,sizeof(Lattice)), "Allocate Memory: lattice_1_device");
	cudasafe(hipMalloc((void **)&lattice_2_device,sizeof(Lattice)), "Allocate Memory: lattice_2_device");
	cudasafe(hipMalloc((void **)&domain_device,sizeof(Domain)), "Allocate Memory: control_device");
	// ARRAYS:
	cudasafe(hipMalloc((void **)&f_1_device,domain_size*Q*sizeof(float)), "Allocate Memory: f_1_device");
	cudasafe(hipMalloc((void **)&f_2_device,domain_size*Q*sizeof(float)), "Allocate Memory: f_2_device");
	cudasafe(hipMalloc((void **)&boundary_type_device,domain_size*sizeof(float)), "Allocate Memory: bounceback_device");
	cudasafe(hipMalloc((void **)&boundary_value_device,domain_size*sizeof(float)), "Allocate Memory: bounceback_device");


}

// READS INPUT DATA FROM FILE AND ASSEMBLES DATA INTO RELEVANT STRUCTS
void load_and_assemble_data(void)
{
	// ASSEMBLE STRUCT ON HOST: Lattice
	lattice_host->f = f_host;

	// ASSEMBLE AND LOAD STRUCT ON HOST: Control
	// ASSEMBLE
	domain_host->boundary_type = boundary_type_host;
	domain_host->boundary_value = boundary_value_host;
	// LOAD
	domain_host->tau = tau;
	domain_host->length.x = length.x;
	domain_host->length.y = length.y;
	domain_host->length.z = length.z;
	
	// Boundary nodes are treated as chains of face nodes, vertex nodes and corner nodes,
	// the length of each of these chains is a function of domain dimensions and is calculated
	// here.
	domain_host->b_o[0] = domain_host->b_o[0]+(length.y-2); // X-
	domain_host->b_o[1] = domain_host->b_o[1]+(length.y-2); // X+
	domain_host->b_o[2] = domain_host->b_o[2]+(length.x-2); // Y-
	domain_host->b_o[3] = domain_host->b_o[3]+(length.x-2); // Y+
	domain_host->b_o[4] = domain_host->b_o[3]+3;
	l_b_o = domain_host->b_o[4];

	// ASSEMBLE STRUCT ON HOST: Output
	output->rho = rho;
	output->ux = ux;
	output->uy = uy;
	output->uz = uz;
	output->u = u;

	// ASSEMBLE STRUCT ON DEVICE: Lattice
	Lattice *lattice_tmp = (Lattice *)malloc(sizeof(Lattice));
	lattice_tmp->f = f_1_device;
	cudasafe(hipMemcpy(lattice_1_device, lattice_tmp, sizeof(Lattice),hipMemcpyHostToDevice),"Copy Data: lattice_1_device");
	lattice_tmp->f = f_2_device;
	cudasafe(hipMemcpy(lattice_2_device, lattice_tmp, sizeof(Lattice),hipMemcpyHostToDevice),"Copy Data: lattice_2_device");

	// ASSEMBLE AND LOAD STRUCT ON DEVICE: Control
	Domain *domain_tmp = (Domain *)malloc(sizeof(Domain));
	domain_tmp->tau = tau;
	domain_tmp->length.x = length.x;
	domain_tmp->length.y = length.y;
	domain_tmp->length.z = length.z;
	domain_tmp->boundary_type = boundary_type_device;
	domain_tmp->boundary_value = boundary_value_device;
	cudasafe(hipMemcpy(domain_device, domain_tmp, sizeof(Domain),hipMemcpyHostToDevice),"Copy Data: control_device");
	cudasafe(hipMemcpy(&domain_device->b_o, &domain_host->b_o, sizeof(int)*19,hipMemcpyHostToDevice),"Copy Data: b_o");
}

// CALCULATES AND LOADS A CONSTANT DENSITY ZERO VELOCITY INITIAL CONDITION FOR THE DOMAIN
void load_static_IC(void)
{
	int index_i;
	float omega[Q];
	LOAD_OMEGA(omega);
	for(int i=0;i<Q;i++)
	{
		for(int index=0;index<(domain_size);index++)
		{
			index_i = index+i*(domain_size);
			lattice_host->f[index_i] = 1.f*omega[i];
		}
	}
	cudasafe(hipMemcpy(f_1_device, f_host, sizeof(float)*Q*domain_size,hipMemcpyHostToDevice),"Copy Data: Initial Condition 1");
	cudasafe(hipMemcpy(f_2_device, f_host, sizeof(float)*Q*domain_size,hipMemcpyHostToDevice),"Copy Data: Initial Condition 2");
}

// EXECUTES ALL ROUTINES REQUIRED FOR THE MODEL SET UP
void setup(void)
{
	FILE * input_file;
    input_file = fopen ("input.dat","r");
	int IC_type, i3d;
	//IC_type = 0;
	fscanf(input_file,"%d %d %d %f %d %d %d", &length.x, &length.y, &length.z, &tau, &saveT, &maxT, &IC_type);
	//printf("%d %d %d %f %d %d %d\n", length.x, length.y, length.z, tau, saveT, maxT, IC_type);
	domain_size = length.x*length.y*length.z;
	allocate_memory_host();
	allocate_memory_device();
	load_and_assemble_data();
	if (IC_type == 0) load_static_IC();
	for (int k = 0; k<length.z; k++)
	{
		for(int j = 0; j<length.y; j++)
		{
			for(int i = 0; i<length.x; i++)
			{
				i3d = i + j*length.x + k*length.x*length.y;
				fscanf(input_file,"%f %f", &domain_host->boundary_type[i3d], &domain_host->boundary_value[i3d]);
				//printf("%f %f\n", domain_host->boundary_type[i3d], domain_host->boundary_value[i3d]);
			}
		}
	}

	cudasafe(hipMemcpy(boundary_type_device, boundary_type_host, sizeof(float)*domain_size,hipMemcpyHostToDevice),"Copy Data: omega_device");
	cudasafe(hipMemcpy(boundary_value_device, boundary_value_host, sizeof(float)*domain_size,hipMemcpyHostToDevice),"Copy Data: omega_device");	
}

// ERROR CHECKING FOR MEMORY ALLOCATION
void cudasafe( hipError_t error, char* message)
{
   if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}

// ERROR CHECKING FOR KERNEL EXECUTION
void Check_CUDA_Error(const char *message)
{
   hipError_t error = hipGetLastError();
   if(error!=hipSuccess) {
      fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
      exit(-1);
   }                         
}

// COPIES f_i DATA FROM DEVICE TO HOST AND COMPUTERS MACROSCOPIC VALUES ON HOST, THIS DATA
// IS THEN WRITTEN TO THE OUTPUT FILE
//
// Note:	A computationally more efficient implementation would compute macroscopic
//			value's on the gpu and then just copy that data, this would however consume
//			more memory
void output_macros(int time)
{
	// Copy data from device to host
	cudasafe(hipMemcpy(f_host, f_1_device, sizeof(float)*Q*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data");
	
	int i3d, target_x, target_y, target_z, ex[Q], ey[Q], ez[Q];
	float rho = 0; float ux = 0; float uy = 0; float uz = 0; float u = 0;
	char fname[19];
	FILE *file;

	LOAD_EX(ex);
	LOAD_EY(ey);
	LOAD_EZ(ez);

// Assemble formatted filename	
	sprintf(fname, "results_%i.dat", time);
// Open file
	file = fopen(fname,"w");
// Write File Header	
	fprintf(file,"TITLE=\"3D Poiseuille Flow\"\nVARIABLES= \"X\", \"Y\", \"Z\", \"rho\", \"uX\", \"uY\", \"uZ\", \"u\"");//\nDATASETAUXDATA ComputerTime=\"%lus\"\nDATASETAUXDATA DeviceMemoryUsed=\"%luMb\"",cputime, mem);
// Write Zone Header
	// note: nx and ny values are not in the "correct" order in the zone header, errors occur when loading the data in tecplot
	// if the "correct" order is used
	fprintf(file,"\nZONE T=\"3D Poiseuille Flow at time = %i\", I=%i, J=%i, K=%i, DATAPACKING=POINT, SOLUTIONTIME=%i", time,length.x,length.y,length.z,time);
// Loop over all nodes to calculate and print nodal macroscopic values to file, output some feedback data to console
	for (int z=0;z<length.z;z++){
		for (int y=0;y<length.y;y++){
			for (int x=0;x<length.x;x++){
				// Calculate macroscopic values
				for(int i =0; i<Q; i++)
				{
					// Streaming occurs prior to collision, therefore we must stream before
					// calculation of macroscopic value's
					target_x = x+ex[i]; target_y = y+ey[i]; target_z = z+ez[i];
					//PERIODIC BOUNDARY
					if(target_x>(length.x-1)) target_x = 1; if(target_x<0) target_x = length.x-1;
					if(target_y>(length.y-1)) target_y = 1; if(target_y<0) target_y = length.y-1;
					if(target_z>(length.z-1)) target_z = 1; if(target_z<0) target_z = length.z-1;

					i3d = (target_x + target_y*length.x + target_z*length.y*length.x)+i*(domain_size);
					rho += lattice_host->f[i3d];
					ux += ex[i]*lattice_host->f[i3d];
					uy += ey[i]*lattice_host->f[i3d];
					uz += ez[i]*lattice_host->f[i3d];
				}

				u = sqrt(ux*ux+uy*uy+uz*uz);
				ux = ux/rho;
				uy = uy/rho;
				uz = uz/rho;

				// Determine which nodes is currently being considered
				int i3d_prime = x+y*length.x+z*length.x*length.y;
				// Impose zero velocity on bounceback nodes
				if(domain_host->boundary_type[i3d_prime] == 0)
				{
					ux = 0;
					uy = 0;
					uz = 0;
					u = 0;
				}
				// Write to files
				fprintf(file,"\n%i %i %i %f %f %f %f %f", x, y, z, rho, ux, uy, uz, u);
				// Output reference information to console
				if (y==length.y/2 && x == length.x/4 && z == length.z/2) {printf("\n time = %i; rho = %f; uX = %f; uY = %f; uZ = %f", time, rho, ux, uy, uz);}
				// Reset macroscopic variable containers
				rho = 0; ux = 0; uy = 0; uz = 0; u = 0;
			}
		}
	}
	// Close file
	fclose(file);
}

// CONFIGURES THE KERNEL CONFIGURATION AND LAUNCHES KERNEL A KERNEL BOTH FOR THE BOUNDARY NODES
// AND THE BULK NODES
//
// Note:	The "all" kernel operates cleanly on both bulk and boundary nodes and may be used
//			instead, though its use is inefficient.
void iterate(void)
{
	// GRID AND BLOCK DEFINITIONS CAN BE CALCULATED BEFORE ITERATE
	// DEFINE BULK GRID AND BLOCK
	dim3 Db_bulk = dim3(length.x-2,1,1);
    dim3 Dg_bulk = dim3(length.y-2,1,1);
	// DEFINE BOUNDARY GRID AND BLOCK
	int boundary_amount = l_b_o;
	int boundary_grid=(int)(boundary_amount/BLOCK_SIZE);
	int boundary_leftover=(boundary_amount%BLOCK_SIZE);

	// ITERATE ONCE
	iterate_bulk_kernel<<<Dg_bulk, Db_bulk>>>(lattice_1_device,lattice_2_device,domain_device);
	Check_CUDA_Error("Kernel \"iterate_bulk 1\" Execution Failed!");  
	iterate_boundary_kernel<<<boundary_grid,BLOCK_SIZE>>>(lattice_1_device,lattice_2_device,domain_device,0);
	Check_CUDA_Error("Kernel \"iterate_boundary 1a\" Execution Failed!");  
	if(boundary_leftover)
		iterate_boundary_kernel<<<1,boundary_leftover>>>(lattice_1_device,lattice_2_device,domain_device,boundary_amount-boundary_leftover);
	Check_CUDA_Error("Kernel \"iterate_boundary 1b\" Execution Failed!");

	// SWAP LATTICES AND ITERATE AGAIN
	iterate_bulk_kernel<<<Dg_bulk, Db_bulk>>>(lattice_2_device,lattice_1_device,domain_device);
	Check_CUDA_Error("Kernel \"iterate_bulk 2\" Execution Failed!");  
	iterate_boundary_kernel<<<boundary_grid,BLOCK_SIZE>>>(lattice_2_device,lattice_1_device,domain_device,0);
	Check_CUDA_Error("Kernel \"iterate_boundary 1a\" Execution Failed!");  
	if(boundary_leftover)
		iterate_boundary_kernel<<<1,boundary_leftover>>>(lattice_2_device,lattice_1_device,domain_device,boundary_amount-boundary_leftover);
	Check_CUDA_Error("Kernel \"iterate_boundary 1b\" Execution Failed!");  
}
/*
void iterate(void)
{
	// GRID AND BLOCK DEFINITIONS CAN BE CALCULATED BEFORE ITERATE
	// DEFINE all GRID AND BLOCK
	dim3 Db_all = dim3(length.x-2,1,1);
    dim3 Dg_all = dim3(length.y-2,length.z-2,1);
	// DEFINE all GRID AND BLOCK
	int all_amount = l_b_o;
	int all_grid=(int)(all_amount/BLOCK_SIZE);
	int all_leftover=(all_amount%BLOCK_SIZE);

	// ITERATE ONCE
	iterate_all_kernel<<<Dg_all, Db_all>>>(lattice_1_device,lattice_2_device,domain_device,0,1);
	Check_CUDA_Error("Kernel \"iterate_all 1\" Execution Failed!");  
	iterate_all_kernel<<<all_grid,BLOCK_SIZE>>>(lattice_1_device,lattice_2_device,domain_device,0,0);
	Check_CUDA_Error("Kernel \"iterate_all 1a\" Execution Failed!");  
	if(all_leftover)
		iterate_all_kernel<<<1,all_leftover>>>(lattice_1_device,lattice_2_device,domain_device,all_amount-all_leftover,0);
	Check_CUDA_Error("Kernel \"iterate_all 1b\" Execution Failed!");

	// SWAP LATTICES AND ITERATE AGAIN
	iterate_all_kernel<<<Dg_all, Db_all>>>(lattice_2_device,lattice_1_device,domain_device,0,1);
	Check_CUDA_Error("Kernel \"iterate_all 2\" Execution Failed!");  
	iterate_all_kernel<<<all_grid,BLOCK_SIZE>>>(lattice_2_device,lattice_1_device,domain_device,0,0);
	Check_CUDA_Error("Kernel \"iterate_all 1a\" Execution Failed!");  
	if(all_leftover)
		iterate_all_kernel<<<1,all_leftover>>>(lattice_2_device,lattice_1_device,domain_device,all_amount-all_leftover,0);
	Check_CUDA_Error("Kernel \"iterate_all 1b\" Execution Failed!");  
}*/

#endif