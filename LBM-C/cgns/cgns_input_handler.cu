#include "hip/hip_runtime.h"
#ifndef CGNS_INPUT_HANDLER
#define CGNS_INPUT_HANDLER

#include <stdio.h>
#include <string>
#include <iostream>
#include <sstream>
#include <vector>
using namespace std;
/* cgnslib.h file must be located in directory specified by -I during compile: */
#include <cgnslib.h>

class CGNSInputHandler
{

    string fname;

    // CGNS variables
    int index_file;
    int length[DIM];

    void open_file()
    {
        cgns_error_check ( cg_open ( fname.c_str(),CG_MODE_READ,&index_file ) );
    }

    void close_file()
    {
        cgns_error_check ( cg_close ( index_file ) );
    }

    void cgns_error_check ( int error_code )
    {
        if ( error_code!=0 )
        {
            const char *error_message = cg_get_error();
            cout << error_message << endl;
            getchar();
            cg_error_exit();
        }
    }

public:
    CGNSInputHandler ( const string &input_filename, int [DIM] );

    CGNSInputHandler ();

    template<class T>
    void read_field ( T *data, char *label )
    {
#warning fix function
        int num_arrays;

        //unused pointers;
        CG_DataType_t d_type;

        bool field_found = false;
        int i;
        char array_name[30];

        cgsize_t min[DIM], max[DIM];

        for ( int i=0; i!=DIM; i++ )
        {
            min[i]=1;
            max[i]=length[i];
        }

        open_file();

        cgns_error_check ( cg_nfields ( index_file, 1, 1, 1, &num_arrays ) );
        for ( i = 1; i<num_arrays+1; i++ )
        {
            cgns_error_check ( cg_field_info ( index_file, 1, 1, 1, i, &d_type, array_name ) );
            if ( strcmp ( array_name, label ) == 0 )
            {
                field_found = true;
                cgns_error_check ( cg_field_info ( index_file, 1, 1, 1, i, &d_type, array_name ) );
                break;
            }
        }
        if ( field_found==true )
        {
            cgns_error_check ( cg_field_read ( index_file, 1, 1, 1, label, d_type, min, max,data ) );
            cout << endl << "Input Handler: " << label << " loaded" << endl;
        }
        else
        {
            cout << endl << "Input Handler: " << label << " not found in file \"" << fname << "\"" << endl;
            exit ( -1 );
        }

        close_file();

    }

};

CGNSInputHandler::CGNSInputHandler ( const string &input_filename, int length_in[DIM] )
{
    fname = input_filename;

    for ( int i=0; i!=DIM; ++i )
        length[i]=length_in[i];

    open_file();

}

CGNSInputHandler::CGNSInputHandler () {}

#endif
