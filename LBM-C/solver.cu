#include "hip/hip_runtime.h"
#ifndef SOLVER
#define SOLVER

#include "solver.cuh"
#include "d2q9_boundary.cu"
#include "collision.cu"

// LIST OF AVAILABLE BOUNDRY TYPES AND COLLISION FUNCTIONS
__device__ boundary_condition boundary_conditions[2] = { zh_pressure_x, zh_pressure_X};
__device__ collision collision_functions[5] = { bgk_collision, guo_bgk_collision, ntpor_collision, guo_ntpor_collision, bounceback};

__global__ void iterate_kernel (Lattice *lattice, DomainArray *domain_arrays, DomainConstant *domain_constants, bool store_macros)
{
	// Declare Variables
	double omega[Q], B;
	int i2d, ex[Q], ey[Q], opp[Q];
	int2 length;
	Node current_node;

	// Initialise variables
	LOAD_EX(ex);
	LOAD_EY(ey);
	LOAD_OMEGA(omega);
	LOAD_OPP(opp);
	current_node.rho = 0; current_node.ux = 0; current_node.uy = 0;
	
	// Compute coordinates
	int x = (blockDim.x*blockIdx.x)+threadIdx.x;
	int y = (blockDim.y*blockIdx.y)+threadIdx.y;

	// Load domain configuration
	length.x = domain_constants->length.x;
	length.y = domain_constants->length.y;
	int domain_size = length.x*length.y;
	double tau = domain_constants->tau;
	int i2d_prime = x + y*length.x;
	
	if(x<length.x && y<length.y)
	{
		// Set collision type and optional forces
		// The type specified in domain_constants must be multiplied by two to match the listing
		// order in the collision_functions array, an additional 1 is added to the collision type
		// to specify a collision with guo body forces
		int collision_modifier = 0;
		if(domain_constants->forcing==true)
		{
			current_node.F[0] = domain_arrays->force[i2d_prime];
			current_node.F[1] = domain_arrays->force[domain_size+i2d_prime];
			if(current_node.F[0] > 0 && current_node.F[1] > 0) collision_modifier = 1;
		}
		int collision_type = (domain_constants->collision_type*2)+collision_modifier;

		// Load boundary condition
		int boundary_type = domain_arrays->boundary_type[i2d_prime];
		double boundary_value = domain_arrays->boundary_value[i2d_prime];
	
		// Load Geometry
		B = domain_arrays->geometry[i2d_prime];
		if(B==1) collision_type = 4;
	
		// STREAMING - UNCOALESCED READ
		int target_x, target_y;
		for(int i = 0; i<Q; i++)
		{
			target_x = x+ex[i]; target_y = y+ey[i];
			//PERIODIC BOUNDARY
			if(target_x>(length.x-1)) target_x = 0; if(target_x<0) target_x = length.x-1;
			if(target_y>(length.y-1)) target_y = 0; if(target_y<0) target_y = length.y-1;
	
			i2d = (target_x + target_y*length.x)+opp[i]*(domain_size);
			
			// UNCOALESCED READ
			current_node.f[opp[i]] = lattice->f_prev[i2d];
	
			current_node.rho += current_node.f[opp[i]];
			current_node.ux += ex[opp[i]]*current_node.f[opp[i]];
			current_node.uy += ey[opp[i]]*current_node.f[opp[i]];
		}
	
		current_node.ux = current_node.ux/current_node.rho;
		current_node.uy = current_node.uy/current_node.rho;
	
		// APPLY BOUNDARY CONDITION
		if (boundary_type>0) boundary_conditions[boundary_type-1](&current_node, &boundary_value);
	
		// COLLISION
		collision_functions[collision_type](&current_node, opp, ex, ey, omega, tau, B);

		// COALESCED WRITE
		__syncthreads();
		for(int i=0;i<Q;i++)
		{
			i2d = (x + y*length.x)+i*(domain_size);
			lattice->f_curr[i2d] = current_node.f[i];
		}

		// STORE MACROS IF REQUIRED
		if (store_macros)
		{
				i2d = (x + y*length.x);
				lattice->ux[i2d] = current_node.ux;
				lattice->uy[i2d] = current_node.uy;
				lattice->u[i2d] = sqrt(current_node.ux*current_node.ux+current_node.uy*current_node.uy);
				lattice->rho[i2d] = current_node.rho;
		} 
	}
}

#endif
