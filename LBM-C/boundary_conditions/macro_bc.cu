#ifndef MACRO_BC
#define MACRO_BC

// Necessary includes
#include "../macros.cu"
#include "macro_bc.cuh"

// These files are only included to remove squiggly red lines in VS2010
#include "../data_types.cuh"
#include "hip/hip_runtime.h"

__device__ __constant__ macro_condition macro_conditions[15] = { presc_ux,		 presc_uy,			presc_uz,
																presc_uy_ux,	 presc_uz_ux,
																presc_uz_uy,	 presc_ux_uy_uz,
																presc_rho,		 presc_rho_ux,		presc_rho_uy,		presc_rho_uz, 
																presc_rho_ux_uy, presc_rho_ux_uz,	presc_rho_uy_uz,
																presc_rho_ux_uy_uz};

__device__ __noinline__ void presc_ux(Node *current_node, Domain *domain)
{
	current_node->u[0] = domain->u[0][current_node->ixd];
}

__device__ __noinline__ void presc_uy(Node *current_node, Domain *domain)
{
	current_node->u[1] = domain->u[1][current_node->ixd];
}

__device__ __noinline__ void presc_uz(Node *current_node, Domain *domain)
{
	#if DIM > 2
		current_node->u[2] = domain->u[2][current_node->ixd];
	#endif
}

__device__ __noinline__ void presc_uy_ux(Node *current_node, Domain *domain)
{
	current_node->u[0] = domain->u[0][current_node->ixd];
	current_node->u[1] = domain->u[1][current_node->ixd];
}

__device__ __noinline__ void presc_uz_ux(Node *current_node, Domain *domain)
{
	current_node->u[0] = domain->u[0][current_node->ixd];
	#if DIM > 2
		current_node->u[2] = domain->u[2][current_node->ixd];
	#endif
}

__device__ __noinline__ void presc_uz_uy(Node *current_node, Domain *domain)
{
	current_node->u[1] = domain->u[1][current_node->ixd];
	#if DIM > 2
		current_node->u[2] = domain->u[2][current_node->ixd];
	#endif
}

__device__ __noinline__ void presc_ux_uy_uz(Node *current_node, Domain *domain)
{
	current_node->u[0] = domain->u[0][current_node->ixd];
	current_node->u[1] = domain->u[1][current_node->ixd];
	#if DIM > 2
		current_node->u[2] = domain->u[2][current_node->ixd];
	#endif
}

__device__ __noinline__ void presc_rho(Node *current_node, Domain *domain)
{
	current_node->rho = domain->rho[current_node->ixd];
}

__device__ __noinline__ void presc_rho_ux(Node *current_node, Domain *domain)
{
	current_node->rho = domain->rho[current_node->ixd];
	current_node->u[0] = domain->u[0][current_node->ixd];
}

__device__ __noinline__ void presc_rho_uy(Node *current_node, Domain *domain)
{
	current_node->rho = domain->rho[current_node->ixd];
	current_node->u[1] = domain->u[1][current_node->ixd];
}

__device__ __noinline__ void presc_rho_uz(Node *current_node, Domain *domain)
{
	current_node->rho = domain->rho[current_node->ixd];
	#if DIM > 2
		current_node->u[2] = domain->u[2][current_node->ixd];
	#endif
}

__device__ __noinline__ void presc_rho_ux_uy(Node *current_node, Domain *domain)
{
	current_node->rho = domain->rho[current_node->ixd];
	current_node->u[0] = domain->u[0][current_node->ixd];
	current_node->u[1] = domain->u[1][current_node->ixd];
}

__device__ __noinline__ void presc_rho_ux_uz(Node *current_node, Domain *domain)
{
	current_node->rho = domain->rho[current_node->ixd];
	current_node->u[0] = domain->u[0][current_node->ixd];
	#if DIM > 2
		current_node->u[2] = domain->u[2][current_node->ixd];
	#endif
}

__device__ __noinline__ void presc_rho_uy_uz(Node *current_node, Domain *domain)
{
	current_node->rho = domain->rho[current_node->ixd];
	current_node->u[1] = domain->u[1][current_node->ixd];
	#if DIM > 2
		current_node->u[2] = domain->u[2][current_node->ixd];
	#endif
}

__device__ __noinline__ void presc_rho_ux_uy_uz(Node *current_node, Domain *domain)
{
	current_node->rho = domain->rho[current_node->ixd];
	current_node->u[0] = domain->u[0][current_node->ixd];
	current_node->u[1] = domain->u[1][current_node->ixd];
	#if DIM > 2
		current_node->u[2] = domain->u[2][current_node->ixd];
	#endif
}
#endif
