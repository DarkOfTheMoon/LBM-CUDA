#include "hip/hip_runtime.h"
#include "data_types.cuh"

//#include <stdio.h>
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <libraryInterfaces/TINYXML_xmlIO.h>
#include <libraryInterfaces/TINYXML_xmlIO.hh>
#include <libraryInterfaces/TINYXML_xmlIO.cpp>
using namespace std;

class InfileReader
{

    DomainConstant *domain_constants;
    Timing *timer;
    OutputController *output_controller;
    ProjectStrings *project;

    void initialise()
    {
        domain_constants->c_smag = 0;
    }

    void parse_file ( plb::XMLreader const& document )
    {
		std::string tmp;
        document["ProjName"].read (project->name );

		std::cout<<project->name<<std::endl;
		exit(0);

        document["DomainFile"].read ( project->domain_fname );
        document["OutputFile"].read ( project->output_fname );
        document["TauMRT"]["TauMRT0"].read ( domain_constants->tau_mrt[0] );
        document["TauMRT"]["TauMRT1"].read ( domain_constants->tau_mrt[1] );
        document["TauMRT"]["TauMRT2"].read ( domain_constants->tau_mrt[2] );
        document["TauMRT"]["TauMRT3"].read ( domain_constants->tau_mrt[3] );
        document["TauMRT"]["TauMRT4"].read ( domain_constants->tau_mrt[4] );
        document["TauMRT"]["TauMRT5"].read ( domain_constants->tau_mrt[5] );
        document["TauMRT"]["TauMRT6"].read ( domain_constants->tau_mrt[6] );
        document["TauMRT"]["TauMRT7"].read ( domain_constants->tau_mrt[7] );
        document["TauMRT"]["TauMRT8"].read ( domain_constants->tau_mrt[8] );
#if DIM>2
        document["TauMRT"]["TauMRT9"].read ( domain_constants->tau_mrt[9] );
        document["TauMRT"]["TauMRT10"].read ( domain_constants->tau_mrt[10] );
        document["TauMRT"]["TauMRT11"].read ( domain_constants->tau_mrt[11] );
        document["TauMRT"]["TauMRT12"].read ( domain_constants->tau_mrt[12] );
        document["TauMRT"]["TauMRT13"].read ( domain_constants->tau_mrt[13] );
        document["TauMRT"]["TauMRT14"].read ( domain_constants->tau_mrt[14] );
#endif
        document["Tau"].read ( domain_constants->tau );

        document["Geometry"]["DeltaX"].read ( domain_constants->h );
        document["Geometry"]["x"].read ( domain_constants->length[0] );
        document["Geometry"]["y"].read ( domain_constants->length[1] );
#if DIM >2
        document["Geometry"]["z"].read ( domain_constants->length[2] );
#endif
        document["DeltaT"].read ( domain_constants->dt );
        document["C_smag"].read ( domain_constants->c_smag );
        //TODO enum{BGK,NTPOR,MRT,MRTPOR}
        document["ColType"].read ( domain_constants->collision_type );

        document["Force"].read ( domain_constants->forcing );
        document["MicroBC"].read ( domain_constants->micro_bc );
        document["MacroBC"].read ( domain_constants->macro_bc );
        document["Tolerance"].read ( domain_constants->tolerance );
        document["Init"].read ( domain_constants->init_type );
        document["MaxT"].read ( timer->max );
        document["FileOut"].read ( timer->plot );
        document["ScreenMes"].read ( timer->screen );

        document["SteadyCheck"].read ( timer->steady_check );

        document["OutputVars"]["u"].read ( output_controller->u[0] );
        document["OutputVars"]["v"].read ( output_controller->u[1] );
#if DIM >2
        document["OutputVars"]["w"].read ( output_controller->u[2] );
#endif
        document["OutputVars"]["rho"].read ( output_controller->rho );
        document["OutputVars"]["pressure"].read ( output_controller->pressure );

        document["ScreenNode"]["x"].read ( output_controller->screen_node[0] );
        document["ScreenNode"]["y"].read ( output_controller->screen_node[1] );
#if DIM >2
        document["ScreenNode"]["z"].read ( output_controller->screen_node[2] );
#endif
        document["Interactive"].read ( output_controller->interactive );

    }

public:
    InfileReader ( plb::XMLreader const& document, ProjectStrings*, DomainConstant *,Timing *,OutputController * );
};

InfileReader::InfileReader ( plb::XMLreader const& document, ProjectStrings *project_in, DomainConstant *domain_constants_in, Timing *timer_in, OutputController *output_controller_in )
{
    project = project_in;
    domain_constants = domain_constants_in;
    timer = timer_in;
    output_controller = output_controller_in;

    cout << endl << "Reading configuration data: " << endl << endl;

    initialise();
    parse_file ( document );

    cout << endl << "Finished reading configuration data." << endl;
}






