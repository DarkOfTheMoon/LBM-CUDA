#include "hip/hip_runtime.h"
#include "data_types.cuh"

//#include <stdio.h>
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include "TINYXML_xmlIO.h"
#include "TINYXML_xmlIO.hh"
using namespace std;

class InfileReader
{

    DomainConstant *domain_constants;
    Timing *timer;
    OutputController *output_controller;
    ProjectStrings *project;

    void initialise()
    {
        domain_constants->c_smag = 0;
    }

    void parse_file ( XMLreader const& document )
    {
        document["Project"]["ProjName"].read ( project->name );
        document["Project"]["DomainFile"].read ( project->domain_fname );
        document["Project"]["OutputFile"].read ( project->output_fname );
        document["Project"]["DomainConst"]["TauMRT"]["TauMRT0"].read ( domain_constants->tau_mrt[0] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT1"].read ( domain_constants->tau_mrt[1] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT2"].read ( domain_constants->tau_mrt[2] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT3"].read ( domain_constants->tau_mrt[3] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT4"].read ( domain_constants->tau_mrt[4] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT5"].read ( domain_constants->tau_mrt[5] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT6"].read ( domain_constants->tau_mrt[6] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT7"].read ( domain_constants->tau_mrt[7] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT8"].read ( domain_constants->tau_mrt[8] );
#if DIM>2
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT9"].read ( domain_constants->tau_mrt[9] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT10"].read ( domain_constants->tau_mrt[10] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT11"].read ( domain_constants->tau_mrt[11] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT12"].read ( domain_constants->tau_mrt[12] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT13"].read ( domain_constants->tau_mrt[13] );
		document["Project"]["DomainConst"]["TauMRT"]["TauMRT14"].read ( domain_constants->tau_mrt[14] );
#endif
		document["Project"]["DomainConst"]["Tau"].read ( domain_constants->tau );

		document["Project"]["DomainConst"]["Geometry"]["DeltaX"].read ( domain_constants->h );
		document["Project"]["DomainConst"]["Geometry"]["x"].read ( domain_constants->length[0] );
		document["Project"]["DomainConst"]["Geometry"]["y"].read ( domain_constants->length[1] );
#if DIM >2
		document["Project"]["DomainConst"]["Geometry"]["z"].read ( domain_constants->length[2] );
#endif
		document["Project"]["DomainConst"]["DeltaT"].read ( domain_constants->dt );
		document["Project"]["DomainConst"]["C_smag"].read ( domain_constants->c_smag );
        //TODO enum{BGK,NTPOR,MRT,MRTPOR}
		document["Project"]["DomainConst"]["ColType"].read ( domain_constants->collision_type );

		document["Project"]["DomainConst"]["Force"].read ( domain_constants->forcing );
		document["Project"]["DomainConst"]["MicroBC"].read ( domain_constants->micro_bc );
		document["Project"]["DomainConst"]["MacroBC"].read ( domain_constants->macro_bc );
		document["Project"]["DomainConst"]["Tolerance"].read ( domain_constants->tolerance );
		document["Project"]["DomainConst"]["Init"].read ( domain_constants->init_type );

        document["Project"]["Timer"]["MaxT"].read ( timer->max );
		document["Project"]["Timer"]["FileOut"].read ( timer->plot );
		document["Project"]["Timer"]["ScreenMes"].read ( timer->screen );
		document["Project"]["Timer"]["SteadyCheck"].read ( timer->steady_check );

        document["Project"]["OutPutController"]["OutputVars"]["u"].read ( output_controller->u[0] );
		document["Project"]["OutPutController"]["OutputVars"]["v"].read ( output_controller->u[1] );
#if DIM >2
		document["Project"]["OutPutController"]["OutputVars"]["w"].read ( output_controller->u[2] );
#endif
		document["Project"]["OutPutController"]["OutputVars"]["rho"].read ( output_controller->rho );
		document["Project"]["OutPutController"]["OutputVars"]["pressure"].read ( output_controller->pressure );

		document["Project"]["OutPutController"]["ScreenNode"]["x"].read ( output_controller->screen_node[0] );
		document["Project"]["OutPutController"]["ScreenNode"]["y"].read ( output_controller->screen_node[1] );
#if DIM >2
		document["Project"]["OutPutController"]["ScreenNode"]["z"].read ( output_controller->screen_node[2] );
#endif
		document["Project"]["OutPutController"]["Interactive"].read ( output_controller->interactive );
    }

public:
    InfileReader ( XMLreader const& document, ProjectStrings*, DomainConstant *,Timing *,OutputController * );
};

InfileReader::InfileReader ( XMLreader const& document, ProjectStrings *project_in, DomainConstant *domain_constants_in, Timing *timer_in, OutputController *output_controller_in )
{
    project = project_in;
    domain_constants = domain_constants_in;
    timer = timer_in;
    output_controller = output_controller_in;

    cout << endl << "Reading configuration data: " << endl << endl;

    initialise();
    parse_file ( document );

    cout << endl << "Finished reading configuration data." << endl;
}







